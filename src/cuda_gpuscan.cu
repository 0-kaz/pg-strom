#include "hip/hip_runtime.h"
/*
 * cuda_gpuscan.cu
 *
 * Device implementation of GpuScan
 * ----
 * Copyright 2011-2023 (C) KaiGai Kohei <kaigai@kaigai.gr.jp>
 * Copyright 2014-2023 (C) PG-Strom Developers Team
 *
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the PostgreSQL License.
 */
#include "cuda_common.h"

/*
 * pgstrom_stair_sum_xxxx
 */
static __shared__ union {
	uint32_t	u32[MAXTHREADS_PER_BLOCK / WARPSIZE];
	uint64_t	u64[MAXTHREADS_PER_BLOCK / WARPSIZE];
	int32_t		i32[MAXTHREADS_PER_BLOCK / WARPSIZE];
	int64_t		i64[MAXTHREADS_PER_BLOCK / WARPSIZE];
	float8_t	fp64[MAXTHREADS_PER_BLOCK / WARPSIZE];
} __stair_sum_buffer;

template <typename T>
INLINE_FUNCTION(T)
__stair_sum_warp_common(T my_value)
{
	T	curr = my_value;
	T	temp;

	assert(__activemask() == ~0U);
	temp = __shfl_sync(__activemask(), curr, (LaneId() & ~0x01));
	if ((LaneId() & 0x01) != 0)
		curr += temp;

	temp = __shfl_sync(__activemask(), curr, (LaneId() & ~0x03) | 0x01);
	if ((LaneId() & 0x02) != 0)
		curr += temp;

	temp = __shfl_sync(__activemask(), curr, (LaneId() & ~0x07) | 0x03);
	if ((LaneId() & 0x04) != 0)
		curr += temp;

	temp = __shfl_sync(__activemask(), curr, (LaneId() & ~0x0f) | 0x07);
	if ((LaneId() & 0x08) != 0)
		curr += temp;

	temp = __shfl_sync(__activemask(), curr, (LaneId() & ~0x1f) | 0x0f);
	if ((LaneId() & 0x10) != 0)
		curr += temp;

	return curr;
}

PUBLIC_FUNCTION(uint32_t)
pgstrom_stair_sum_binary(bool predicate, uint32_t *p_total_count)
{
	uint32_t	n_warps = get_local_size() / warpSize;
	uint32_t	warp_id = get_local_id()   / warpSize;
	uint32_t	mask;
	uint32_t	sum;

	assert(__activemask() == ~0U);
	mask = __ballot_sync(__activemask(), predicate);
	if (LaneId() == 0)
		__stair_sum_buffer.u32[warp_id] = __popc(mask);
	__syncthreads();

	if (warp_id == 0)
	{
		uint32_t	temp = (LaneId() < n_warps ? __stair_sum_buffer.u32[LaneId()] : 0);

		__stair_sum_buffer.u32[LaneId()] = __stair_sum_warp_common(temp);
	}
	__syncthreads();

	if (p_total_count)
		*p_total_count = __stair_sum_buffer.u32[warpSize-1];
	sum = (warp_id > 0 ? __stair_sum_buffer.u32[warp_id-1] : 0);
	__syncthreads();

	mask &= ((1U << LaneId()) - 1);		/* not include myself */
	return sum + __popc(mask);
}

#define PGSTROM_STAIR_SUM_TEMPLATE(SUFFIX, BASETYPE, FIELD)				\
	PUBLIC_FUNCTION(BASETYPE)											\
	pgstrom_stair_sum_##SUFFIX(BASETYPE value, BASETYPE *p_total_count)	\
	{																	\
		uint32_t	n_warps = get_local_size() / warpSize;				\
		uint32_t	warp_id = get_local_id()   / warpSize;				\
		BASETYPE	warp_sum;											\
		BASETYPE	sum;												\
																		\
		assert(__activemask() == ~0U);									\
		warp_sum = __stair_sum_warp_common(value);						\
		assert(warp_sum >= value);										\
		if (LaneId() == warpSize - 1)									\
			__stair_sum_buffer.FIELD[warp_id] = warp_sum;				\
		__syncthreads();												\
																		\
		if (warp_id == 0)												\
		{																\
			BASETYPE	temp = (LaneId() < n_warps						\
								? __stair_sum_buffer.FIELD[LaneId()] : 0); \
			__stair_sum_buffer.FIELD[LaneId()] = __stair_sum_warp_common(temp);	\
		}																\
		__syncthreads();												\
																		\
		if (p_total_count)												\
			*p_total_count = __stair_sum_buffer.FIELD[warpSize-1];		\
		sum = (warp_id > 0 ? __stair_sum_buffer.FIELD[warp_id-1] : 0);	\
		__syncthreads();												\
																		\
		return sum + warp_sum;											\
	}

PGSTROM_STAIR_SUM_TEMPLATE(uint32, uint32_t, u32)
PGSTROM_STAIR_SUM_TEMPLATE(uint64, uint64_t, u64)
PGSTROM_STAIR_SUM_TEMPLATE(int64,  int64_t,  i64)
PGSTROM_STAIR_SUM_TEMPLATE(fp64,   float8_t, fp64)

#define PGSTROM_LOCAL_MINMAX_TEMPLATE(SUFFIX, BASETYPE, FIELD, OPER, INVAL)	\
	PUBLIC_FUNCTION(BASETYPE)											\
	pgstrom_local_##SUFFIX(BASETYPE my_value)							\
	{																	\
		int			warp_id = get_local_id()   / warpSize;				\
		int			n_warps = get_local_size() / warpSize;				\
		BASETYPE	curr = my_value;									\
		BASETYPE	temp;												\
																		\
		/* makes warp local min/max */									\
		assert(__activemask() == ~0U);									\
		temp = __shfl_xor_sync(__activemask(), curr, 0x0001);			\
		curr = OPER(curr, temp);										\
		temp = __shfl_xor_sync(__activemask(), curr, 0x0002);			\
		curr = OPER(curr, temp);										\
		temp = __shfl_xor_sync(__activemask(), curr, 0x0004);			\
		curr = OPER(curr, temp);										\
		temp = __shfl_xor_sync(__activemask(), curr, 0x0008);			\
		curr = OPER(curr, temp);										\
		temp = __shfl_xor_sync(__activemask(), curr, 0x0010);			\
		curr = OPER(curr, temp);										\
																		\
		if (LaneId() == 0)												\
			__stair_sum_buffer.FIELD[warp_id] = curr;					\
		__syncthreads();												\
																		\
		if (warp_id == 0)												\
		{																\
			assert(__activemask() == ~0U);								\
			curr = (LaneId() < n_warps ? __stair_sum_buffer.FIELD[LaneId()] : INVAL); \
																		\
			temp = __shfl_xor_sync(__activemask(), curr, 0x0001);		\
			curr = OPER(curr, temp);									\
			temp = __shfl_xor_sync(__activemask(), curr, 0x0002);		\
			curr = OPER(curr, temp);									\
			temp = __shfl_xor_sync(__activemask(), curr, 0x0004);		\
			curr = OPER(curr, temp);									\
			temp = __shfl_xor_sync(__activemask(), curr, 0x0008);		\
			curr = OPER(curr, temp);									\
			temp = __shfl_xor_sync(__activemask(), curr, 0x0010);		\
			curr = OPER(curr, temp);									\
																		\
			__stair_sum_buffer.FIELD[LaneId()] = curr;					\
		}																\
		__syncthreads();												\
		curr = __stair_sum_buffer.FIELD[LaneId()];						\
		__syncthreads();												\
		return curr;													\
	}

PGSTROM_LOCAL_MINMAX_TEMPLATE(min_int32, int32_t, i32,  Min,  INT_MAX)
PGSTROM_LOCAL_MINMAX_TEMPLATE(max_int32, int32_t, i32,  Max,  INT_MIN)
PGSTROM_LOCAL_MINMAX_TEMPLATE(min_int64, int64_t, i64,  Min,  LONG_MAX)
PGSTROM_LOCAL_MINMAX_TEMPLATE(max_int64, int64_t, i64,  Max,  LONG_MIN)
PGSTROM_LOCAL_MINMAX_TEMPLATE(min_fp64, float8_t, fp64, Min,  DBL_MAX)
PGSTROM_LOCAL_MINMAX_TEMPLATE(max_fp64, float8_t, fp64, Max, -DBL_MAX)

/* ----------------------------------------------------------------
 *
 * execGpuScanLoadSource and related
 *
 * ----------------------------------------------------------------
 */
STATIC_FUNCTION(int)
__gpuscan_load_source_row(kern_context *kcxt,
						  kern_warp_context *wp,
						  const kern_data_store *kds_src,
						  const kern_expression *kexp_load_vars,
						  const kern_expression *kexp_scan_quals,
						  const kern_expression *kexp_move_vars,
						  char *dst_kvecs_buffer)
{
	uint32_t	count;
	uint32_t	index;
	uint32_t	wr_pos;
	kern_tupitem *tupitem = NULL;

	/* compute the next row-index */
	count = wp->smx_row_count;
	__syncthreads();
	if (get_local_id() == 0)
		wp->smx_row_count++;
	index = get_global_size() * count + get_global_base();
	if (index >= kds_src->nitems)
	{
		if (get_local_id() == 0)
			wp->scan_done = 1;
		__syncthreads();
		return 1;
	}
	index += get_local_id();

	/*
	 * fetch the outer tuple to scan
	 */
	if (index < kds_src->nitems)
	{
		uint32_t	offset = KDS_GET_ROWINDEX(kds_src)[index];

		assert(offset <= kds_src->usage);
		tupitem = (kern_tupitem *)((char *)kds_src +
								   kds_src->length -
								   __kds_unpack(offset));
		assert((char *)tupitem >= (char *)kds_src &&
			   (char *)tupitem <  (char *)kds_src + kds_src->length);
		if (!ExecLoadVarsOuterRow(kcxt,
								  kexp_load_vars,
								  kexp_scan_quals,
								  kds_src,
								  &tupitem->htup))
			tupitem = NULL;
	}
	/* error checks */
	if (__syncthreads_count(kcxt->errcode != ERRCODE_STROM_SUCCESS) > 0)
		return -1;

	/*
	 * save the private kvars slot on the combination buffer (depth=0)
	 */
	wr_pos = WARP_WRITE_POS(wp,0);
	wr_pos += pgstrom_stair_sum_binary(tupitem != NULL, &count);
	if (get_local_id() == 0)
		WARP_WRITE_POS(wp,0) += count;
	if (tupitem != NULL)
	{
		if (!ExecMoveKernelVariables(kcxt,
									 kexp_move_vars,
									 dst_kvecs_buffer,
									 (wr_pos % KVEC_UNITSZ)))
		{
			assert(kcxt->errcode != ERRCODE_STROM_SUCCESS);
		}
	}
	/* error checks */
	if (__syncthreads_count(kcxt->errcode != ERRCODE_STROM_SUCCESS) > 0)
		return -1;
	/* move to the next depth, if more than blockSize tuples were fetched. */
	return (WARP_WRITE_POS(wp,0) >= WARP_READ_POS(wp,0) + get_local_size() ? 1 : 0);
}

/*
 * __gpuscan_load_source_block
 */
STATIC_FUNCTION(int)
__gpuscan_load_source_block(kern_context *kcxt,
							kern_warp_context *wp,
							const kern_data_store *kds_src,
							const kern_expression *kexp_load_vars,
							const kern_expression *kexp_scan_quals,
							const kern_expression *kexp_move_vars,
							char *dst_kvecs_buffer)
{
	uint32_t	wr_pos = wp->lp_wr_pos;
	uint32_t	rd_pos = wp->lp_rd_pos;
	uint32_t	block_id;
	uint32_t	count;
	bool		has_next_lp_items = false;
	HeapTupleHeaderData *htup = NULL;

	assert(wr_pos >= rd_pos);
	block_id = (get_global_size() / warpSize) * wp->smx_row_count;
	if (block_id >= kds_src->nitems || wr_pos >= rd_pos + get_local_size())
	{
		uint32_t	off;

		rd_pos += get_local_id();
		if (rd_pos < wr_pos)
		{
			off = wp->lp_items[rd_pos % LP_ITEMS_PER_BLOCK];
			htup = (HeapTupleHeaderData *)((char *)kds_src + __kds_unpack(off));
			if (!ExecLoadVarsOuterRow(kcxt,
									  kexp_load_vars,
									  kexp_scan_quals,
									  kds_src, htup))
				htup = NULL;
		}
		/* error checks */
		if (__syncthreads_count(kcxt->errcode != ERRCODE_STROM_SUCCESS) > 0)
			return -1;
		if (get_local_id() == 0)
			wp->lp_rd_pos = Min(wp->lp_wr_pos,
								wp->lp_rd_pos + get_local_size());
		/*
		 * save the private kvars on the warp-buffer
		 */
		wr_pos = WARP_WRITE_POS(wp,0);
		wr_pos += pgstrom_stair_sum_binary(htup != NULL, &count);
		if (get_local_id() == 0)
			WARP_WRITE_POS(wp,0) += count;
		if (htup != NULL)
		{
			if (!ExecMoveKernelVariables(kcxt,
										 kexp_move_vars,
										 dst_kvecs_buffer,
										 (wr_pos % KVEC_UNITSZ)))
			{
				assert(kcxt->errcode != ERRCODE_STROM_SUCCESS);
			}
		}
		/* error checks */
		if (__syncthreads_count(kcxt->errcode != ERRCODE_STROM_SUCCESS) > 0)
			return -1;
		/* end-of-scan checks */
		if (block_id >= kds_src->nitems &&	/* no more blocks to fetch */
			wp->lp_rd_pos >= wp->lp_wr_pos)	/* no more pending tuples */
		{
			if (get_local_id() == 0)
				wp->scan_done = 1;
			return 1;
		}
		/* move to the next depth if more than blockSize tuples were fetched */
		return (WARP_WRITE_POS(wp,0) >= WARP_READ_POS(wp,0) + get_local_size() ? 1 : 0);
	}

	/*
	 * Here, number of pending tuples (which is saved in the lp_items[]) is
	 * not enough to run ScanQuals checks. So, we move to the next bunch of
	 * line-items or next block.
	 * The pending tuples just passed the MVCC visivility checks, but
	 * ScanQuals check is not applied yet. We try to run ScanQuals checks
	 * with maximum number of threads simultaneously, as large as we can.
	 */
	block_id += (get_global_id() / warpSize);
	if (block_id < kds_src->nitems)
	{
		PageHeaderData *pg_page = KDS_BLOCK_PGPAGE(kds_src, block_id);
		BlockNumber		block_nr = KDS_BLOCK_BLCKNR(kds_src, block_id);
		uint32_t		nitems = PageGetMaxOffsetNumber(pg_page);
		uint32_t		index;

		index = wp->lp_count * warpSize + LaneId();
		if (index < PageGetMaxOffsetNumber(pg_page))
		{
			ItemIdData *lpp = &pg_page->pd_linp[index];

			assert((char *)lpp < (char *)pg_page + BLCKSZ);
			if (ItemIdIsNormal(lpp))
			{
				htup = (HeapTupleHeaderData *)PageGetItem(pg_page, lpp);
				/* for ctid system column reference */
				htup->t_ctid.ip_blkid.bi_hi = (uint16_t)(block_nr >> 16);
				htup->t_ctid.ip_blkid.bi_lo = (uint16_t)(block_nr & 0xffffU);
				htup->t_ctid.ip_posid = index + 1;
			}
		}
		has_next_lp_items = (index + warpSize < nitems);
	}
	/* put visible tuples on the lp_items[] array */
	wr_pos = wp->lp_wr_pos;
	wr_pos += pgstrom_stair_sum_binary(htup != NULL, &count);
	if (get_local_id() == 0)
		wp->lp_wr_pos += count;
	if (htup != NULL)
	{
		wp->lp_items[wr_pos % LP_ITEMS_PER_BLOCK]
			= __kds_packed((char *)htup - (char *)kds_src);
	}
	/* increment the row/line pointer */
	if (__syncthreads_count(has_next_lp_items) > 0)
	{
		if (get_local_id() == 0)
			wp->lp_count++;
	}
	else
	{
		if (get_local_id() == 0)
		{
			wp->smx_row_count++;
			wp->lp_count = 0;
		}
	}
	return 0;	/* stay depth=0 */
}

/*
 * __gpuscan_load_source_arrow
 */
STATIC_FUNCTION(int)
__gpuscan_load_source_arrow(kern_context *kcxt,
							kern_warp_context *wp,
							const kern_data_store *kds_src,
							const kern_expression *kexp_load_vars,
							const kern_expression *kexp_scan_quals,
							const kern_expression *kexp_move_vars,
							char *dst_kvecs_buffer)
{
	uint32_t	count;
	uint32_t	index;
	uint32_t	wr_pos;
	bool		is_valid = false;

	/* compute the next row-index */
	count = wp->smx_row_count;
	__syncthreads();
	if (get_local_id() == 0)
		wp->smx_row_count++;
	index = get_global_size() * count + get_global_base();
	if (index >= kds_src->nitems)
	{
		if (get_local_id() == 0)
			wp->scan_done = 1;
		return 1;
	}
	index += get_local_id();

	/*
	 * fetch arrow tuple
	 */
	if (index < kds_src->nitems)
	{
		if (ExecLoadVarsOuterArrow(kcxt,
								   kexp_load_vars,
								   kexp_scan_quals,
								   kds_src,
								   index))
			is_valid = true;
	}
	/* error checks */
    if (__syncthreads_count(kcxt->errcode != ERRCODE_STROM_SUCCESS) > 0)
		return -1;
	/*
	 * save the private kvars slot on the combination buffer (depth=0)
	 */
	wr_pos = WARP_WRITE_POS(wp,0);
	wr_pos += pgstrom_stair_sum_binary(is_valid, &count);
	if (get_local_id() == 0)
		WARP_WRITE_POS(wp,0) += count;
	if (is_valid)
	{
		if (!ExecMoveKernelVariables(kcxt,
									 kexp_move_vars,
									 dst_kvecs_buffer,
									 (wr_pos % KVEC_UNITSZ)))
		{
			assert(kcxt->errcode != ERRCODE_STROM_SUCCESS);
		}
	}
	/* error checks */
	if (__syncthreads_count(kcxt->errcode != ERRCODE_STROM_SUCCESS) > 0)
		return -1;
	/* move to the next depth, if more than blockSize rows were fetched. */
	return (WARP_WRITE_POS(wp,0) >= WARP_READ_POS(wp,0) + get_local_size() ? 1 : 0);
}

/*
 * __gpuscan_load_source_column (KDS_FORMAT_COLUMN)
 */
INLINE_FUNCTION(GpuCacheSysattr *)
kds_column_get_sysattr(const kern_data_store *kds, uint32_t rowid)
{
	const kern_colmeta *cmeta = &kds->colmeta[kds->nr_colmeta - 1];
	GpuCacheSysattr	   *base;

	assert(!cmeta->attbyval &&
		   cmeta->attalign == sizeof(uint32_t) &&
		   cmeta->attlen == sizeof(GpuCacheSysattr) &&
		   cmeta->nullmap_offset == 0);
	base = (GpuCacheSysattr *)
		((char *)kds + __kds_unpack(cmeta->values_offset));
	if (rowid < kds->column_nrooms)
		return &base[rowid];
	return NULL;
}

STATIC_FUNCTION(bool)
kds_column_check_visibility(kern_context *kcxt,
							const kern_data_store *kds,
							uint32_t rowid)
{
	SerializedTransactionState *xstate = SESSION_XACT_STATE(kcxt->session);
	GpuCacheSysattr *sysattr = kds_column_get_sysattr(kds, rowid);

	assert(xstate != NULL && sysattr != NULL);

	if (sysattr->xmin == InvalidTransactionId)
		return false;
	if (sysattr->xmin != FrozenTransactionId)
	{
		for (int i=0; i < xstate->nParallelCurrentXids; i++)
		{
			if (sysattr->xmin == xstate->parallelCurrentXids[i])
				goto xmin_is_visible;
		}
		return false;
	}
xmin_is_visible:
	if (sysattr->xmax == InvalidTransactionId)
		return true;
	if (sysattr->xmax == FrozenTransactionId)
		return false;
	for (int i=0; i < xstate->nParallelCurrentXids; i++)
	{
		if (sysattr->xmax == xstate->parallelCurrentXids[i])
			return false;
	}
	return true;
}

STATIC_FUNCTION(int)
__gpuscan_load_source_column(kern_context *kcxt,
							 kern_warp_context *wp,
							 const kern_data_store *kds_src,
							 const kern_data_extra *kds_extra,
							 const kern_expression *kexp_load_vars,
							 const kern_expression *kexp_scan_quals,
							 const kern_expression *kexp_move_vars,
							 char *dst_kvecs_buffer)
{
	uint32_t	count;
	uint32_t	index;
	uint32_t	wr_pos;
	bool		is_valid = false;

	/* fetch next blockSize tuples */
	count = wp->smx_row_count;
	__syncthreads();
	if (get_local_id() == 0)
		wp->smx_row_count++;
	index = get_global_size() * count + get_global_base();
	if (index >= kds_src->nitems)
	{
		if (get_local_id() == 0)
			wp->scan_done = 1;
		return 1;
	}
	index += get_local_id();

	/*
	 * fetch the outer tuple to scan
	 */
	if (index < kds_src->nitems &&
		kds_column_check_visibility(kcxt, kds_src, index))
	{
		if (ExecLoadVarsOuterColumn(kcxt,
									kexp_load_vars,
									kexp_scan_quals,
									kds_src,
									kds_extra,
									index))
			is_valid = true;
	}
	/* error checks */
	if (__syncthreads_count(kcxt->errcode != ERRCODE_STROM_SUCCESS) > 0)
		return -1;
	/*
	 * save the private kvars slot on the combination buffer (depth=0)
	 */
	wr_pos = WARP_WRITE_POS(wp,0);
	wr_pos += pgstrom_stair_sum_binary(is_valid, &count);
	if (get_local_id() == 0)
		WARP_WRITE_POS(wp,0) += count;
	if (is_valid)
	{
		if (!ExecMoveKernelVariables(kcxt,
									 kexp_move_vars,
									 dst_kvecs_buffer,
									 (wr_pos % KVEC_UNITSZ)))
		{
			assert(kcxt->errcode != ERRCODE_STROM_SUCCESS);
		}
	}
	/* error checks */
	if (__syncthreads_count(kcxt->errcode != ERRCODE_STROM_SUCCESS) > 0)
        return -1;
	/* move to the next depth if more than 32 htuples were fetched */
	return (WARP_WRITE_POS(wp,0) >= WARP_READ_POS(wp,0) + get_local_size() ? 1 : 0);
}

PUBLIC_FUNCTION(int)
execGpuScanLoadSource(kern_context *kcxt,
					  kern_warp_context *wp,
					  const kern_data_store *kds_src,
					  const kern_data_extra *kds_extra,
					  const kern_expression *kexp_load_vars,
					  const kern_expression *kexp_scan_quals,
					  const kern_expression *kexp_move_vars,
					  char *dst_kvecs_buffer)
{
	/*
	 * Move to the next depth (or projection), if combination buffer (depth=0)
	 * may overflow on the next action, or we already reached to the KDS tail.
	 */
	if (wp->scan_done > 0 ||
		WARP_WRITE_POS(wp,0) >= WARP_READ_POS(wp,0) + get_local_size())
		return 1;

	/* no source kernel-vectorized buffer for depth==0 */
	kcxt->kvecs_curr_buffer = NULL;
	kcxt->kvecs_curr_id = 0;

	switch (kds_src->format)
	{
		case KDS_FORMAT_ROW:
			return __gpuscan_load_source_row(kcxt, wp,
											 kds_src,
											 kexp_load_vars,
											 kexp_scan_quals,
											 kexp_move_vars,
											 dst_kvecs_buffer);
		case KDS_FORMAT_BLOCK:
			return __gpuscan_load_source_block(kcxt, wp,
											   kds_src,
											   kexp_load_vars,
											   kexp_scan_quals,
											   kexp_move_vars,
											   dst_kvecs_buffer);
		case KDS_FORMAT_ARROW:
			return __gpuscan_load_source_arrow(kcxt, wp,
											   kds_src,
											   kexp_load_vars,
											   kexp_scan_quals,
											   kexp_move_vars,
											   dst_kvecs_buffer);
		case KDS_FORMAT_COLUMN:
			return __gpuscan_load_source_column(kcxt, wp,
												kds_src,
												kds_extra,
												kexp_load_vars,
												kexp_scan_quals,
												kexp_move_vars,
												dst_kvecs_buffer);
		default:
			STROM_ELOG(kcxt, "Bug? Unknown KDS format");
			break;
	}
	return -1;
}

/* ------------------------------------------------------------
 *
 * Routines to manage GpuCache
 *
 * ------------------------------------------------------------
 */
STATIC_FUNCTION(void)
gpucache_cleanup_row_owner(kern_context *kcxt,
						   kern_gpucache_redolog *redo,
						   kern_data_store *kds)
{
	uint32_t	owner_id;

	for (owner_id = get_global_id();
		 owner_id < redo->nitems;
		 owner_id += get_global_size())
	{
		GCacheTxLogCommon *tx_log;
		GpuCacheSysattr *sysattr;
		uint32_t		offset = redo->redo_items[owner_id];
		uint32_t		rowid;

		tx_log = (GCacheTxLogCommon *)
			((char *)redo + __kds_unpack(offset));
		switch (tx_log->type)
		{
			case GCACHE_TX_LOG__INSERT:
				rowid = ((GCacheTxLogInsert *)tx_log)->rowid;
				break;
			case GCACHE_TX_LOG__DELETE:
				rowid = ((GCacheTxLogDelete *)tx_log)->rowid;
				break;
			case GCACHE_TX_LOG__COMMIT_INS:
			case GCACHE_TX_LOG__COMMIT_DEL:
			case GCACHE_TX_LOG__ABORT_INS:
			case GCACHE_TX_LOG__ABORT_DEL:
				rowid = ((GCacheTxLogXact *)tx_log)->rowid;
				break;
			default:
				STROM_ELOG(kcxt, "unknown GCacheTxLog type");
				return;
		}
		assert(rowid < kds->column_nrooms);
		sysattr = kds_column_get_sysattr(kds, rowid);
		sysattr->owner = 0;
	}
}

STATIC_FUNCTION(void)
gpucache_assign_update_owner(kern_context *kcxt,
							 kern_gpucache_redolog *redo,
							 kern_data_store *kds)
{
	uint32_t	owner_id;

	for (owner_id = get_global_id();
		 owner_id < redo->nitems;
		 owner_id += get_global_size())
	{
		GCacheTxLogCommon *tx_log;
		GpuCacheSysattr *sysattr;
		uint32_t		offset = redo->redo_items[owner_id];
		uint32_t		rowid;

		tx_log = (GCacheTxLogCommon *)
			((char *)redo + __kds_unpack(offset));
		if (tx_log->type == GCACHE_TX_LOG__INSERT)
		{
			rowid = ((GCacheTxLogInsert *)tx_log)->rowid;
			sysattr = kds_column_get_sysattr(kds, rowid);
			__atomic_max_uint32(&sysattr->owner, owner_id);
		}
		else if (tx_log->type == GCACHE_TX_LOG__DELETE)
		{
			rowid = ((GCacheTxLogDelete *)tx_log)->rowid;
			sysattr = kds_column_get_sysattr(kds, rowid);
			__atomic_max_uint32(&sysattr->owner, owner_id);			
		}
	}
}

STATIC_FUNCTION(bool)
__gpucache_apply_insert_log(kern_context *kcxt,
							kern_data_store *kds,
							kern_data_extra *extra,
							GpuCacheSysattr *sysattr,
							const GCacheTxLogInsert *i_log)
{
	const HeapTupleHeaderData *htup = &i_log->htup;
	uint32_t	rowid = i_log->rowid;
	bool		heap_hasnull = ((htup->t_infomask & HEAP_HASNULL) != 0);
	uint32_t	offset = htup->t_hoff;
	int			j, ncols = Min(kds->ncols, (htup->t_infomask2 & HEAP_NATTS_MASK));

	for (j=0; j < ncols; j++)
	{
		const kern_colmeta *cmeta = &kds->colmeta[j];
		char	   *base;

		if (cmeta->nullmap_offset != 0)
		{
			uint32_t   *nullmap = (uint32_t *)
				((char *)kds + __kds_unpack(cmeta->nullmap_offset));

			if (heap_hasnull && att_isnull(j, htup->t_bits))
			{
				__atomic_and_uint32(&nullmap[rowid>>5], ~(1U<<(rowid&31)));
				continue;
			}
			else
			{
				__atomic_or_uint32(&nullmap[rowid>>5], (1U<<(rowid&31)));
			}
		}
		else
		{
			if (heap_hasnull && att_isnull(j, htup->t_bits))
			{
				STROM_ELOG(kcxt, "NULL appeared at not-null column");
				return false;
			}
		}

		assert(cmeta->values_offset != 0);
		base = (char *)kds + __kds_unpack(cmeta->values_offset);
		if (cmeta->attlen > 0)
		{
			offset = TYPEALIGN(cmeta->attalign, offset);
			memcpy(base + cmeta->attlen * rowid,
				   (char *)htup + offset,
				   cmeta->attlen);
			offset += cmeta->attlen;
		}
		else
		{
			char	   *vl_pos;
			uint32_t	vl_len;
			uint32_t	vl_off;

			assert(cmeta->attlen == -1);
			if (!VARATT_NOT_PAD_BYTE((char *)htup + offset))
				offset = TYPEALIGN(cmeta->attalign, offset);
			vl_pos = (char *)htup + offset;
			vl_len = VARSIZE_ANY(vl_pos);
			vl_off = __atomic_add_uint64(&extra->usage, MAXALIGN(vl_len));
			if (vl_off + vl_len > extra->length)
			{
				STROM_EREPORT(kcxt, ERRCODE_BUFFER_NO_SPACE,
							  "gpucache: extra buffer has no space");
				return false;
			}
			memcpy((char *)extra + vl_off,
				   (char *)htup + offset,
				   vl_len);
			((uint32_t *)base)[rowid] = __kds_packed(vl_off);
			offset += vl_len;
		}
	}
	sysattr->xmin = htup->t_choice.t_heap.t_xmin;
	sysattr->xmax = htup->t_choice.t_heap.t_xmax;
	memcpy(&sysattr->ctid, &htup->t_ctid, sizeof(ItemPointerData));
	
	return true;
}

STATIC_FUNCTION(void)
gpucache_apply_update_logs(kern_context *kcxt,
						   kern_gpucache_redolog *redo,
						   kern_data_store *kds,
						   kern_data_extra *extra)
{
	__shared__ uint32_t smx_rowid_max;
	uint32_t	rowid_max = UINT_MAX;
	uint32_t	owner_id;

	if (get_local_id() == 0)
		smx_rowid_max = 0;
	__syncthreads();

	for (owner_id = get_global_id();
		 owner_id < redo->nitems;
		 owner_id += get_global_size())
	{
		GCacheTxLogCommon *tx_log;
		GpuCacheSysattr *sysattr;
		uint32_t		offset = redo->redo_items[owner_id];

		tx_log = (GCacheTxLogCommon *)
			((char *)redo + __kds_unpack(offset));
		if (tx_log->type == GCACHE_TX_LOG__INSERT)
		{
			GCacheTxLogInsert  *i_log = (GCacheTxLogInsert *)tx_log;

			sysattr = kds_column_get_sysattr(kds, i_log->rowid);
			if (sysattr->owner == owner_id)
			{
				__gpucache_apply_insert_log(kcxt, kds, extra, sysattr, i_log);
				if (rowid_max == UINT_MAX || rowid_max < i_log->rowid)
					rowid_max = i_log->rowid;
			}
		}
		else if (tx_log->type == GCACHE_TX_LOG__DELETE)
		{
			GCacheTxLogDelete  *d_log = (GCacheTxLogDelete *)tx_log;

			sysattr = kds_column_get_sysattr(kds, d_log->rowid);
			if (sysattr->owner == owner_id)
			{
				sysattr->xmax = d_log->xid;
				if (rowid_max == UINT_MAX || rowid_max < d_log->rowid)
					rowid_max = d_log->rowid;
			}
		}
	}
	/* update kds->nitems */
	if (rowid_max != UINT_MAX)
		__atomic_max_uint32(&smx_rowid_max, rowid_max);
	if (__syncthreads_count(rowid_max != UINT_MAX) > 0 && get_local_id() == 0)
		__atomic_max_uint32(&kds->nitems, smx_rowid_max+1);
}

STATIC_FUNCTION(void)
gpucache_assign_xact_owner(kern_context *kcxt,
							 kern_gpucache_redolog *redo,
							 kern_data_store *kds)
{
	uint32_t	owner_id;

	for (owner_id = get_global_id();
		 owner_id < redo->nitems;
		 owner_id += get_global_size())
	{
		GCacheTxLogCommon *tx_log;
		GpuCacheSysattr *sysattr;
		uint32_t		offset = redo->redo_items[owner_id];
		uint32_t		rowid;

		tx_log = (GCacheTxLogCommon *)
			((char *)redo + __kds_unpack(offset));
		if (tx_log->type == GCACHE_TX_LOG__COMMIT_INS ||
			tx_log->type == GCACHE_TX_LOG__COMMIT_DEL ||
			tx_log->type == GCACHE_TX_LOG__ABORT_INS ||
			tx_log->type == GCACHE_TX_LOG__ABORT_DEL)
		{
			rowid = ((GCacheTxLogXact *)tx_log)->rowid;
			 sysattr = kds_column_get_sysattr(kds, rowid);
            __atomic_max_uint32(&sysattr->owner, owner_id);
		}
	}
}

STATIC_FUNCTION(uint64_t)
__gpucache_count_deadspace(kern_data_store *kds,
						   kern_data_extra *extra,
						   uint32_t rowid)
{
	uint64_t	retval = 0;

	if (kds->has_varlena)
	{
		assert(rowid < kds->column_nrooms);
		for (int j=0; j < kds->ncols; j++)
		{
			const kern_colmeta *cmeta = &kds->colmeta[j];

			if (cmeta->attlen > 0)
				continue;
			assert(cmeta->attlen == -1);
			if (!KDS_COLUMN_ITEM_ISNULL(kds, cmeta, rowid))
			{
				uint32_t   *base = (uint32_t *)
					((char *)kds + __kds_unpack(cmeta->values_offset));
				char	   *vl = (char *)extra + __kds_unpack(base[rowid]);

				retval += MAXALIGN(VARSIZE_ANY(vl));
			}
		}
	}
	return retval;
}


STATIC_FUNCTION(void)
gpucache_apply_xact_logs(kern_context *kcxt,
						 kern_gpucache_redolog *redo,
						 kern_data_store *kds,
						 kern_data_extra *extra)
{
	__shared__ uint32_t smx_rowid_max;
	__shared__ uint64_t smx_deadspace;
	uint32_t	rowid_max = UINT_MAX;
	uint32_t	owner_id;
	uint64_t	sz;

	if (get_local_id() == 0)
	{
		smx_rowid_max = 0;
		smx_deadspace = 0;
	}
	__syncthreads();

	for (owner_id = get_global_id();
		 owner_id < redo->nitems;
		 owner_id += get_global_size())
	{
		GCacheTxLogXact *tx_log;
		GpuCacheSysattr *sysattr;
		uint32_t		offset = redo->redo_items[owner_id];

		tx_log = (GCacheTxLogXact *)
			((char *)redo + __kds_unpack(offset));
		switch (tx_log->type)
		{
			case GCACHE_TX_LOG__COMMIT_INS:
				sysattr = kds_column_get_sysattr(kds, tx_log->rowid);
				if (sysattr->owner == owner_id)
				{
					sysattr->xmin = FrozenTransactionId;
					if (rowid_max == UINT_MAX || rowid_max < tx_log->rowid)
						rowid_max = tx_log->rowid;
				}
				break;
			case GCACHE_TX_LOG__COMMIT_DEL:
				sysattr = kds_column_get_sysattr(kds, tx_log->rowid);
				if (sysattr->owner == owner_id)
				{
					sysattr->xmax = FrozenTransactionId;
					if (rowid_max == UINT_MAX || rowid_max < tx_log->rowid)
						rowid_max = tx_log->rowid;
					sz = __gpucache_count_deadspace(kds, extra, tx_log->rowid);
					if (sz > 0)
						__atomic_add_uint64(&smx_deadspace, sz);
				}
				break;
			case GCACHE_TX_LOG__ABORT_INS:
				sysattr = kds_column_get_sysattr(kds, tx_log->rowid);
				if (sysattr->owner == owner_id)
				{
					sysattr->xmin = InvalidTransactionId;
					if (rowid_max == UINT_MAX || rowid_max < tx_log->rowid)
						rowid_max = tx_log->rowid;
					sz = __gpucache_count_deadspace(kds, extra, tx_log->rowid);
					if (sz > 0)
						__atomic_add_uint64(&smx_deadspace, sz);
				}
				break;
			case GCACHE_TX_LOG__ABORT_DEL:
				sysattr = kds_column_get_sysattr(kds, tx_log->rowid);
				if (sysattr->owner == owner_id)
				{
					sysattr->xmax = InvalidTransactionId;
					if (rowid_max == UINT_MAX || rowid_max < tx_log->rowid)
						rowid_max = tx_log->rowid;
				}
				break;
			default:
				break;
		}
	}
	/* update kds->nitems */
	if (rowid_max != UINT_MAX)
		__atomic_max_uint32(&smx_rowid_max, rowid_max);
	if (__syncthreads_count(rowid_max != UINT_MAX) > 0 && get_local_id() == 0)
		__atomic_max_uint32(&kds->nitems, smx_rowid_max+1);
	/* update extra->deadspace */
	if (get_local_id() == 0 && smx_deadspace > 0)
	{
		assert(extra != NULL);
		__atomic_add_uint64(&extra->deadspace, smx_deadspace);
	}
}

KERNEL_FUNCTION(void)
kern_gpucache_apply_redo(kern_gpucache_redolog *gcache_redo,
						 kern_data_store *kds,
						 kern_data_extra *extra,
						 int phase)
{
	kern_context	kcxt;	/* just for error message */

	/* bailout if any errors */
	if (__syncthreads_count(gcache_redo->kerror.errcode) > 0)
		return;

	memset(&kcxt, 0, offsetof(kern_context, vlbuf));
	switch (phase)
	{
		case 1:		/* clean up the owner_id of sysattr */
			gpucache_cleanup_row_owner(&kcxt, gcache_redo, kds);
			break;
		case 2:		/* assign the largest owner_id of INS/DEL log entries */
			gpucache_assign_update_owner(&kcxt, gcache_redo, kds);
			break;
		case 3:		/* apply INS/DEL log entries */
			gpucache_apply_update_logs(&kcxt, gcache_redo, kds, extra);
			break;
		case 4:		/* clean up the owner_id of sysattr */
			gpucache_cleanup_row_owner(&kcxt, gcache_redo, kds);
            break;
		case 5:		/* assign the largest owner_id of XACT log entries */
			gpucache_assign_xact_owner(&kcxt, gcache_redo, kds);
			break;
		case 6:		/* apply XACT log entries */
			gpucache_apply_xact_logs(&kcxt, gcache_redo, kds, extra);
			break;
		default:
			STROM_ELOG(&kcxt, "gpucache: unknown phase");
			break;
	}
	STROM_WRITEBACK_ERROR_STATUS(&gcache_redo->kerror, &kcxt);
}

KERNEL_FUNCTION(void)
kern_gpucache_compaction(kern_data_store *kds,
						 kern_data_extra *extra_src,
						 kern_data_extra *extra_dst)
{
	uint32_t	index;

	for (index = get_global_id();
		 index < kds->nitems;
		 index += get_global_size())
	{
		for (int j=0; j < kds->ncols; j++)
		{
			kern_colmeta   *cmeta = &kds->colmeta[j];
			uint32_t	   *values;
			char		   *vl_src;
			uint32_t		vl_len;
			uint64_t		offset;

			if (cmeta->attlen >= 0)
				continue;
			if (cmeta->nullmap_offset != 0)
			{
				uint8_t	   *nullmap = (uint8_t *)
					((char *)kds + __kds_unpack(cmeta->nullmap_offset));

				if (att_isnull(index, nullmap))
					continue;
			}
			values = (uint32_t *)
				((char *)kds + __kds_unpack(cmeta->values_offset));
			vl_src = ((char *)extra_src + __kds_unpack(values[index]));
			vl_len = VARSIZE_ANY(vl_src);

			offset = __atomic_add_uint64(&extra_dst->usage, MAXALIGN(vl_len));
			if (offset + vl_len <= extra_dst->length)
			{
				memcpy((char *)extra_dst + offset, vl_src, vl_len);
				values[index] = __kds_packed(offset);
			}
		}
	}
}
