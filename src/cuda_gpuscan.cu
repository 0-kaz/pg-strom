#include "hip/hip_runtime.h"
/*
 * cuda_gpuscan.cu
 *
 * Device implementation of GpuScan
 * ----
 * Copyright 2011-2023 (C) KaiGai Kohei <kaigai@kaigai.gr.jp>
 * Copyright 2014-2023 (C) PG-Strom Developers Team
 *
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the PostgreSQL License.
 */
#include "cuda_common.h"

/* ----------------------------------------------------------------
 *
 * execGpuScanLoadSource and related
 *
 * ----------------------------------------------------------------
 */
STATIC_FUNCTION(int)
__gpuscan_load_source_row(kern_context *kcxt,
						  kern_warp_context *wp,
						  kern_data_store *kds_src,
						  kern_expression *kexp_load_vars,
						  kern_expression *kexp_scan_quals,
						  char *kvars_addr_wp,
						  uint32_t *p_smx_row_count)
{
	uint32_t	count;
	uint32_t	index;
	uint32_t	mask;
	uint32_t	wr_pos;
	kern_tupitem *tupitem = NULL;

	/* fetch next warpSize tuples */
	if (LaneId() == 0)
		count = atomicAdd(p_smx_row_count, 1);
	count = __shfl_sync(__activemask(), count, 0);
	index = (get_num_groups() * count + get_group_id()) * warpSize;
	if (index >= kds_src->nitems)
	{
		if (LaneId() == 0)
			wp->scan_done = 1;
		__syncwarp();
		return 1;
	}
	index += LaneId();

	if (index < kds_src->nitems)
	{
		uint32_t	offset = KDS_GET_ROWINDEX(kds_src)[index];

		assert(offset <= kds_src->usage);
		tupitem = (kern_tupitem *)((char *)kds_src +
								   kds_src->length -
								   __kds_unpack(offset));
		assert((char *)tupitem >= (char *)kds_src &&
			   (char *)tupitem <  (char *)kds_src + kds_src->length);
		kcxt->kvars_slot = (kern_variable *)alloca(kcxt->kvars_nbytes);
		kcxt->kvars_class = (int *)(kcxt->kvars_slot + kcxt->kvars_nslots);
		if (!ExecLoadVarsOuterRow(kcxt,
								  kexp_load_vars,
								  kexp_scan_quals,
								  kds_src,
								  &tupitem->htup))
			tupitem = NULL;
	}
	/* error checks */
	if (__any_sync(__activemask(), kcxt->errcode != ERRCODE_STROM_SUCCESS))
		return -1;
	/*
	 * save the private kvars slot on the combination buffer (depth=0)
	 */
	mask = __ballot_sync(__activemask(), tupitem != NULL);
	if (LaneId() == 0)
	{
		wr_pos = WARP_WRITE_POS(wp,0);
		WARP_WRITE_POS(wp,0) += __popc(mask);
	}
	wr_pos = __shfl_sync(__activemask(), wr_pos, 0);
	mask &= ((1U << LaneId()) - 1);
	wr_pos += __popc(mask);
	if (tupitem != NULL)
	{
		index = (wr_pos % UNIT_TUPLES_PER_DEPTH);
		memcpy((char *)kvars_addr_wp + index * kcxt->kvars_nbytes,
			   kcxt->kvars_slot,
			   kcxt->kvars_nbytes);
	}
	kcxt->kvars_slot = NULL;
	kcxt->kvars_class = NULL;
	__syncwarp();
	/* move to the next depth if more than 32 htuples were fetched */
	return (WARP_WRITE_POS(wp,0) >= WARP_READ_POS(wp,0) + warpSize ? 1 : 0);
}

/*
 * __gpuscan_load_source_block
 */
STATIC_FUNCTION(int)
__gpuscan_load_source_block(kern_context *kcxt,
							kern_warp_context *wp,
							kern_data_store *kds_src,
							kern_expression *kexp_load_vars,
							kern_expression *kexp_scan_quals,
							char *kvars_addr_wp,
							uint32_t *p_smx_row_count)
{
	uint32_t	block_id = __shfl_sync(__activemask(), wp->block_id, 0);
	uint32_t	wr_pos = __shfl_sync(__activemask(), wp->lp_wr_pos, 0);
	uint32_t	rd_pos = __shfl_sync(__activemask(), wp->lp_rd_pos, 0);
	uint32_t	count;
	uint32_t	mask;

	assert(wr_pos >= rd_pos);
	if (block_id > kds_src->nitems || wr_pos >= rd_pos + warpSize)
	{
		HeapTupleHeaderData *htup = NULL;
		uint32_t	off;
		int			index;

		rd_pos += LaneId();
		if (rd_pos < wr_pos)
		{
			off = wp->lp_items[rd_pos % UNIT_TUPLES_PER_DEPTH];
			htup = (HeapTupleHeaderData *)((char *)kds_src + __kds_unpack(off));
			kcxt->kvars_slot = (kern_variable *)alloca(kcxt->kvars_nbytes);
			kcxt->kvars_class = (int *)(kcxt->kvars_slot + kcxt->kvars_nslots);
			if (!ExecLoadVarsOuterRow(kcxt,
									  kexp_load_vars,
									  kexp_scan_quals,
									  kds_src, htup))
				htup = NULL;
		}
		/* error checks */
		if (__any_sync(__activemask(), kcxt->errcode != ERRCODE_STROM_SUCCESS))
			return -1;
		if (LaneId() == 0)
			wp->lp_rd_pos = Min(wp->lp_wr_pos,
								wp->lp_rd_pos + warpSize);
		/*
		 * save the private kvars on the warp-buffer
		 */
		mask = __ballot_sync(__activemask(), htup != NULL);
		if (LaneId() == 0)
		{
			wr_pos = WARP_WRITE_POS(wp,0);
			WARP_WRITE_POS(wp,0) += __popc(mask);
		}
		wr_pos = __shfl_sync(__activemask(), wr_pos, 0);
		mask &= ((1U << LaneId()) - 1);
		wr_pos += __popc(mask);
		if (htup != NULL)
		{
			index = (wr_pos % UNIT_TUPLES_PER_DEPTH);
			memcpy(kvars_addr_wp + index * kcxt->kvars_nbytes,
				   kcxt->kvars_slot,
				   kcxt->kvars_nbytes);
		}
		kcxt->kvars_slot = NULL;
		kcxt->kvars_class = NULL;
		__syncwarp();
		/* end-of-scan checks */
		if (block_id > kds_src->nitems &&	/* no more blocks to fetch */
			wp->lp_rd_pos >= wp->lp_wr_pos)	/* no more pending tuples  */
		{
			if (LaneId() == 0)
				wp->scan_done = 1;
			return 1;
		}
		/* move to the next depth if more than 32 htuples were fetched */
		return (WARP_WRITE_POS(wp,0) >= WARP_READ_POS(wp,0) + warpSize ? 1 : 0);
	}

	/*
	 * Here, number of pending tuples (which is saved in the lp_items[]) is
	 * not enough to run ScanQuals checks. So, we move to the next bunch of
	 * line-items or next block.
	 * The pending tuples just passed the MVCC visivility checks, but
	 * ScanQuals check is not applied yet. We try to run ScanQuals checks
	 * with 32 threads simultaneously.
	 */
	if (block_id == 0)
	{
		/*
		 * block_id == 0 means this warp is not associated with particular
		 * block-page, so we try to fetch the next page.
		 */
		if (LaneId() == 0)
			count = atomicAdd(p_smx_row_count, 1);
		count = __shfl_sync(__activemask(), count, 0);
		block_id = (get_num_groups() * count + get_group_id()) + 1;
		if (LaneId() == 0)
			wp->block_id = block_id;
	}
	if (block_id <= kds_src->nitems)
	{
		HeapTupleHeaderData *htup = NULL;
		PageHeaderData *pg_page = KDS_BLOCK_PGPAGE(kds_src, block_id-1);
		BlockNumber		block_nr = KDS_BLOCK_BLCKNR(kds_src, block_id-1);

		count = __shfl_sync(__activemask(), wp->lp_count, 0);
		if (count < PageGetMaxOffsetNumber(pg_page))
		{
			count += LaneId();
			if (count < PageGetMaxOffsetNumber(pg_page))
			{
				ItemIdData *lpp = &pg_page->pd_linp[count];

				assert((char *)lpp < (char *)pg_page + BLCKSZ);
				if (ItemIdIsNormal(lpp))
				{
					htup = (HeapTupleHeaderData *)PageGetItem(pg_page, lpp);
					/* for ctid system column reference */
					htup->t_ctid.ip_blkid.bi_hi = (uint16_t)(block_nr >> 16);
					htup->t_ctid.ip_blkid.bi_lo = (uint16_t)(block_nr & 0xffffU);
					htup->t_ctid.ip_posid = count + 1;
				}
				else
					htup = NULL;
			}
			/* put visible tuples on the lp_items[] array */
			mask = __ballot_sync(__activemask(), htup != NULL);
			if (LaneId() == 0)
			{
				wr_pos = wp->lp_wr_pos;
				wp->lp_wr_pos += __popc(mask);
			}
			wr_pos = __shfl_sync(__activemask(), wr_pos, 0);
			mask &= ((1U << LaneId()) - 1);
			wr_pos += __popc(mask);
			if (htup != NULL)
			{
				wp->lp_items[wr_pos % UNIT_TUPLES_PER_DEPTH]
					= __kds_packed((char *)htup - (char *)kds_src);
			}
			if (LaneId() == 0)
				wp->lp_count += warpSize;
		}
		else
		{
			/* no more tuples to fetch from the current page */
			if (LaneId() == 0)
			{
				wp->block_id = 0;
				wp->lp_count = 0;
			}
			__syncwarp();
		}
	}
	return 0;	/* stay depth-0 */
}

/*
 * __gpuscan_load_source_arrow
 */
STATIC_FUNCTION(int)
__gpuscan_load_source_arrow(kern_context *kcxt,
							kern_warp_context *wp,
							kern_data_store *kds_src,
							kern_expression *kexp_load_vars,
							kern_expression *kexp_scan_quals,
							char *kvars_addr_wp,
							uint32_t *p_smx_row_count)
{
	uint32_t	kds_index;
	uint32_t	count;
	uint32_t	mask;
	uint32_t	wr_pos;
	bool		is_valid = false;

	/* fetch next warpSize tuples */
	if (LaneId() == 0)
		count = atomicAdd(p_smx_row_count, 1);
	count = __shfl_sync(__activemask(), count, 0);
	kds_index = (get_num_groups() * count + get_group_id()) * warpSize;
	if (kds_index >= kds_src->nitems)
	{
		wp->scan_done = 1;
		__syncwarp(__activemask());
		return 1;
	}
	kds_index += LaneId();

	if (kds_index < kds_src->nitems)
	{
		kcxt->kvars_slot = (kern_variable *)alloca(kcxt->kvars_nbytes);
		kcxt->kvars_class = (int *)(kcxt->kvars_slot + kcxt->kvars_nslots);
		if (ExecLoadVarsOuterArrow(kcxt,
								   kexp_load_vars,
								   kexp_scan_quals,
								   kds_src,
								   kds_index))
			is_valid = true;
	}
	/* error checks */
	if (__any_sync(__activemask(), kcxt->errcode != 0))
		return -1;
	/*
	 * save the htuple on the local combination buffer (depth=0)
	 */
	mask = __ballot_sync(__activemask(), is_valid);
	if (LaneId() == 0)
	{
		wr_pos = WARP_WRITE_POS(wp,0);
		WARP_WRITE_POS(wp,0) += __popc(mask);
	}
	wr_pos = __shfl_sync(__activemask(), wr_pos, 0);
	mask &= ((1U << LaneId()) - 1);
	wr_pos += __popc(mask);
	if (is_valid)
	{
		int		index = (wr_pos % UNIT_TUPLES_PER_DEPTH);

		memcpy(kvars_addr_wp + index * kcxt->kvars_nbytes,
			   kcxt->kvars_slot,
			   kcxt->kvars_nbytes);
	}
	kcxt->kvars_slot = NULL;
	kcxt->kvars_class = NULL;
	/* move to the next depth if more than 32 htuples were fetched */
	return (WARP_WRITE_POS(wp,0) >= WARP_READ_POS(wp,0) + warpSize ? 1 : 0);
}

/*
 * __gpuscan_load_source_column
 */
STATIC_FUNCTION(int)
__gpuscan_load_source_column(kern_context *kcxt,
							 kern_warp_context *wp,
							 kern_data_store *kds_src,
							 kern_data_extra *kds_extra,
							 kern_expression *kexp_load_vars,
							 kern_expression *kexp_scan_quals,
							 char *kvars_addr_wp,
							 uint32_t *p_smx_row_count)
{
	uint32_t	count;
	uint32_t	index;
	uint32_t	mask;
	uint32_t	wr_pos;
	bool		row_is_valid = false;

	/* fetch next warpSize tuples */
	if (LaneId() == 0)
		count = atomicAdd(p_smx_row_count, 1);
	count = __shfl_sync(__activemask(), count, 0);
	index = (get_num_groups() * count + get_group_id()) * warpSize;
	if (index >= kds_src->nitems)
	{
		if (LaneId() == 0)
			wp->scan_done = 1;
		__syncwarp();
		return 1;
	}
	index += LaneId();

	if (index < kds_src->nitems)
	{
		kcxt->kvars_slot = (kern_variable *)alloca(kcxt->kvars_nbytes);
		kcxt->kvars_class = (int *)(kcxt->kvars_slot + kcxt->kvars_nslots);
		if (!ExecLoadVarsOuterColumn(kcxt,
									 kexp_load_vars,
									 kexp_scan_quals,
									 kds_src,
									 kds_extra,
									 index))
			row_is_valid = true;
	}
	/* error checks */
	if (__any_sync(__activemask(), kcxt->errcode != ERRCODE_STROM_SUCCESS))
		return -1;
	/*
	 * save the private kvars slot on the combination buffer (depth=0)
	 */
	mask = __ballot_sync(__activemask(), row_is_valid);
	if (LaneId() == 0)
	{
		wr_pos = WARP_WRITE_POS(wp,0);
		WARP_WRITE_POS(wp,0) += __popc(mask);
	}
	wr_pos = __shfl_sync(__activemask(), wr_pos, 0);
	mask &= ((1U << LaneId()) - 1);
	wr_pos += __popc(mask);
	if (row_is_valid)
	{
		index = (wr_pos % UNIT_TUPLES_PER_DEPTH);
		memcpy((char *)kvars_addr_wp + index * kcxt->kvars_nbytes,
			   kcxt->kvars_slot,
			   kcxt->kvars_nbytes);
	}
	kcxt->kvars_slot = NULL;
	kcxt->kvars_class = NULL;
	__syncwarp();
	/* move to the next depth if more than 32 htuples were fetched */
	return (WARP_WRITE_POS(wp,0) >= WARP_READ_POS(wp,0) + warpSize ? 1 : 0);
}

PUBLIC_FUNCTION(int)
execGpuScanLoadSource(kern_context *kcxt,
					  kern_warp_context *wp,
					  kern_data_store *kds_src,
					  kern_data_extra *kds_extra,
					  kern_expression *kexp_load_vars,
					  kern_expression *kexp_scan_quals,
					  char *kvars_addr_wp,
					  uint32_t *p_smx_row_count)
{
	/*
	 * Move to the next depth (or projection), if combination buffer (depth=0)
	 * may overflow on the next action, or we already reached to the KDS tail.
	 */
	if (wp->scan_done || WARP_WRITE_POS(wp,0) >= WARP_READ_POS(wp,0) + warpSize)
		return 1;

	switch (kds_src->format)
	{
		case KDS_FORMAT_ROW:
			return __gpuscan_load_source_row(kcxt, wp,
											 kds_src,
											 kexp_load_vars,
											 kexp_scan_quals,
											 kvars_addr_wp,
											 p_smx_row_count);
		case KDS_FORMAT_BLOCK:
			return __gpuscan_load_source_block(kcxt, wp,
											   kds_src,
											   kexp_load_vars,
											   kexp_scan_quals,
											   kvars_addr_wp,
											   p_smx_row_count);
		case KDS_FORMAT_ARROW:
			return __gpuscan_load_source_arrow(kcxt, wp,
											   kds_src,
											   kexp_load_vars,
											   kexp_scan_quals,
											   kvars_addr_wp,
											   p_smx_row_count);
		case KDS_FORMAT_COLUMN:
			return __gpuscan_load_source_column(kcxt, wp,
												kds_src,
												kds_extra,
												kexp_load_vars,
												kexp_scan_quals,
												kvars_addr_wp,
												p_smx_row_count);
		default:
			STROM_ELOG(kcxt, "Bug? Unknown KDS format");
			break;
	}
	return -1;
}

/* ------------------------------------------------------------
 *
 * Routines to manage GpuCache
 *
 * ------------------------------------------------------------
 */
INLINE_FUNCTION(GpuCacheSysattr *)
kds_column_get_sysattr(kern_data_store *kds, uint32_t rowid)
{
	const kern_colmeta *cmeta = &kds->colmeta[kds->nr_colmeta - 1];
	GpuCacheSysattr	   *base;

	assert(!cmeta->attbyval &&
		   cmeta->attalign == sizeof(uint32_t) &&
		   cmeta->attlen == sizeof(GpuCacheSysattr) &&
		   cmeta->nullmap_offset == 0);
	base = (GpuCacheSysattr *)
		((char *)kds + __kds_unpack(cmeta->values_offset));
	if (rowid < kds->column_nrooms)
		return &base[rowid];
	return NULL;
}

STATIC_FUNCTION(void)
gpucache_cleanup_row_owner(kern_context *kcxt,
						   kern_gpucache_redolog *redo,
						   kern_data_store *kds)
{
	uint32_t	owner_id;

	for (owner_id = get_global_id();
		 owner_id < redo->nitems;
		 owner_id += get_global_size())
	{
		GCacheTxLogCommon *tx_log;
		GpuCacheSysattr *sysattr;
		uint32_t		offset = redo->redo_items[owner_id];
		uint32_t		rowid;

		tx_log = (GCacheTxLogCommon *)
			((char *)redo + __kds_unpack(offset));
		if (tx_log->type == GCACHE_TX_LOG__MAGIC)
		{
			rowid = ((GCacheTxLogInsert *)tx_log)->rowid;
		}
		else if (tx_log->type == GCACHE_TX_LOG__DELETE)
		{
			rowid = ((GCacheTxLogDelete *)tx_log)->rowid;
		}
		else if (tx_log->type == GCACHE_TX_LOG__COMMIT_INS ||
				 tx_log->type == GCACHE_TX_LOG__COMMIT_DEL ||
				 tx_log->type == GCACHE_TX_LOG__ABORT_INS ||
				 tx_log->type == GCACHE_TX_LOG__ABORT_DEL)
		{
			rowid = ((GCacheTxLogXact *)tx_log)->rowid;
		}
		else
		{
			STROM_ELOG(kcxt, "unknown GCacheTxLog type");
			break;
		}
		assert(rowid < kds->column_nrooms);
		sysattr = kds_column_get_sysattr(kds, rowid);
		sysattr->owner = 0;
	}
}

STATIC_FUNCTION(void)
gpucache_assign_update_owner(kern_context *kcxt,
							 kern_gpucache_redolog *redo,
							 kern_data_store *kds)
{
	uint32_t	owner_id;

	for (owner_id = get_global_id();
		 owner_id < redo->nitems;
		 owner_id += get_global_size())
	{
		GCacheTxLogCommon *tx_log;
		GpuCacheSysattr *sysattr;
		uint32_t		offset = redo->redo_items[owner_id];
		uint32_t		rowid;

		tx_log = (GCacheTxLogCommon *)
			((char *)redo + __kds_unpack(offset));
		if (tx_log->type == GCACHE_TX_LOG__MAGIC)
		{
			rowid = ((GCacheTxLogInsert *)tx_log)->rowid;
			sysattr = kds_column_get_sysattr(kds, rowid);
			__atomic_max_uint32(&sysattr->owner, owner_id);
		}
		else if (tx_log->type == GCACHE_TX_LOG__DELETE)
		{
			rowid = ((GCacheTxLogDelete *)tx_log)->rowid;
			sysattr = kds_column_get_sysattr(kds, rowid);
			__atomic_max_uint32(&sysattr->owner, owner_id);			
		}
	}
}

STATIC_FUNCTION(void)
gpucache_apply_update_logs(kern_context *kcxt,
						   kern_gpucache_redolog *redo,
						   kern_data_store *kds,
						   kern_data_extra *extra)
{}

STATIC_FUNCTION(void)
gpucache_assign_xact_owner(kern_context *kcxt,
							 kern_gpucache_redolog *redo,
							 kern_data_store *kds)
{
	uint32_t	owner_id;

	for (owner_id = get_global_id();
		 owner_id < redo->nitems;
		 owner_id += get_global_size())
	{
		GCacheTxLogCommon *tx_log;
		GpuCacheSysattr *sysattr;
		uint32_t		offset = redo->redo_items[owner_id];
		uint32_t		rowid;

		tx_log = (GCacheTxLogCommon *)
			((char *)redo + __kds_unpack(offset));
		if (tx_log->type == GCACHE_TX_LOG__COMMIT_INS ||
			tx_log->type == GCACHE_TX_LOG__COMMIT_DEL ||
			tx_log->type == GCACHE_TX_LOG__ABORT_INS ||
			tx_log->type == GCACHE_TX_LOG__ABORT_DEL)
		{
			rowid = ((GCacheTxLogXact *)tx_log)->rowid;
			 sysattr = kds_column_get_sysattr(kds, rowid);
            __atomic_max_uint32(&sysattr->owner, owner_id);
		}
	}
}

STATIC_FUNCTION(void)
gpucache_apply_xact_logs(kern_context *kcxt,
						 kern_gpucache_redolog *redo,
						 kern_data_store *kds,
						 kern_data_extra *extra)
{
	uint32_t	owner_id;

	for (owner_id = get_global_id();
		 owner_id < redo->nitems;
		 owner_id += get_global_size())
	{
		GCacheTxLogXact *tx_log;
		GpuCacheSysattr *sysattr;
		uint32_t		offset = redo->redo_items[owner_id];
		uint32_t		rowid;

		tx_log = (GCacheTxLogXact *)
			((char *)redo + __kds_unpack(offset));
		switch (tx_log->type)
		{
			case GCACHE_TX_LOG__COMMIT_INS:
				sysattr = kds_column_get_sysattr(kds, tx_log->rowid);
				//do commit insert
				break;
			case GCACHE_TX_LOG__COMMIT_DEL:
				sysattr = kds_column_get_sysattr(kds, tx_log->rowid);
				//do commit delete
				break;
			case GCACHE_TX_LOG__ABORT_INS:
				sysattr = kds_column_get_sysattr(kds, tx_log->rowid);
				//do abort insert
				break;
			case GCACHE_TX_LOG__ABORT_DEL:
				sysattr = kds_column_get_sysattr(kds, tx_log->rowid);
//				if (sysattr->owner_id == owner_id)
//					sysattr->xmax = tx_log->xid;
				//do abort delete
				break;
			default:
				break;
		}
	}
}

KERNEL_FUNCTION(void)
kern_gpucache_apply_redo(kern_gpucache_redolog *gcache_redo,
						 kern_data_store *kds,
						 kern_data_extra *extra,
						 int phase)
{
	kern_context	kcxt;	/* just for error message */

	/* bailout if any errors */
	if (__syncthreads_count(gcache_redo->kerror.errcode) > 0)
		return;

	memset(&kcxt, 0, offsetof(kern_context, vlbuf));
	switch (phase)
	{
		case 1:		/* clean up the owner_id of sysattr */
			gpucache_cleanup_row_owner(&kcxt, gcache_redo, kds);
			break;
		case 2:		/* assign the largest owner_id of INS/DEL log entries */
			gpucache_assign_update_owner(&kcxt, gcache_redo, kds);
			break;
		case 3:		/* apply INS/DEL log entries */
			gpucache_apply_update_logs(&kcxt, gcache_redo, kds, extra);
			break;
		case 4:		/* clean up the owner_id of sysattr */
			gpucache_cleanup_row_owner(&kcxt, gcache_redo, kds);
            break;
		case 5:		/* assign the largest owner_id of XACT log entries */
			gpucache_assign_xact_owner(&kcxt, gcache_redo, kds);
			break;
		case 6:		/* apply XACT log entries */
			gpucache_apply_xact_logs(&kcxt, gcache_redo, kds, extra);
			break;
		default:
			STROM_ELOG(&kcxt, "gpucache: unknown phase");
			break;
	}
	STROM_WRITEBACK_ERROR_STATUS(&gcache_redo->kerror, &kcxt);
}

KERNEL_FUNCTION(void)
kern_gpucache_compaction(kern_data_store *kds,
						 kern_data_extra *extra_src,
						 kern_data_extra *extra_dst)
{
	uint32_t	index;

	for (index = get_global_id();
		 index < kds->nitems;
		 index += get_global_size())
	{
		for (int j=0; j < kds->ncols; j++)
		{
			kern_colmeta   *cmeta = &kds->colmeta[j];
			uint32_t	   *values;
			char		   *vl_src;
			uint32_t		vl_len;
			uint64_t		offset;

			if (cmeta->attlen >= 0)
				continue;
			if (cmeta->nullmap_offset != 0)
			{
				uint8_t	   *nullmap = (uint8_t *)
					((char *)kds + __kds_unpack(cmeta->nullmap_offset));

				if (att_isnull(index, nullmap))
					continue;
			}
			values = (uint32_t *)
				((char *)kds + __kds_unpack(cmeta->values_offset));
			vl_src = ((char *)extra_src + __kds_unpack(values[index]));
			vl_len = VARSIZE_ANY(vl_src);

			offset = __atomic_add_uint64(&extra_dst->usage, MAXALIGN(vl_len));
			if (offset + vl_len <= extra_dst->length)
			{
				memcpy((char *)extra_dst + offset, vl_src, vl_len);
				values[index] = __kds_packed(offset);
			}
		}
	}
}
