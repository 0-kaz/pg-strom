#include "hip/hip_runtime.h"
/*
 * cuda_gpujoin.cu
 *
 * GPU accelerated parallel relations join based on hash-join or
 * nested-loop logic.
 * --
 * Copyright 2011-2023 (C) KaiGai Kohei <kaigai@kaigai.gr.jp>
 * Copyright 2014-2023 (C) PG-Strom Developers Team
 *
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the PostgreSQL License.
 */
#include "cuda_common.h"

/*
 * GPU Nested-Loop
 */
STATIC_FUNCTION(int)
execGpuJoinNestLoop(kern_context *kcxt,
					kern_warp_context *wp,
					kern_multirels *kmrels,
					int			depth,
					char	   *src_kvars_addr_wp,
					char	   *dst_kvars_addr_wp,
					uint32_t   &l_state,
					bool	   &matched)
{
	kern_data_store *kds_heap = KERN_MULTIRELS_INNER_KDS(kmrels, depth-1);
	bool	   *oj_map = KERN_MULTIRELS_OUTER_JOIN_MAP(kmrels, depth-1);
	kern_expression *kexp;
	uint32_t	read_pos;
	uint32_t	write_pos;
	uint32_t	mask;
	bool		tuple_is_valid = false;

	if (WARP_WRITE_POS(wp,depth) >= WARP_READ_POS(wp,depth) + warpSize)
	{
		/*
		 * The destination depth already keeps warpSize or more pending
		 * tuple. So, flush out these tuples first.
		 */
		return depth+1;
	}

	if (__all_sync(__activemask(), l_state >= kds_heap->nitems))
	{
		/*
		 * OK, all the threads in this warp reached to the end of hash-slot
		 * chain. Due to the above checks, the next depth has enough space
		 * to store the result in this depth.
		 */
		if (LaneId() == 0)
			WARP_READ_POS(wp,depth-1) = Min(WARP_READ_POS(wp,depth-1) + warpSize,
											WARP_WRITE_POS(wp,depth-1));
		__syncwarp();
		l_state = 0;
		matched = false;
		if (wp->scan_done >= depth)
		{
			assert(wp->scan_done == depth);
			if (WARP_READ_POS(wp,depth-1) >= WARP_WRITE_POS(wp,depth-1))
			{
				if (LaneId() == 0)
					wp->scan_done = depth + 1;
				return depth+1;
			}
			/*
			 * Elsewhere, remaining tuples in the combination buffer
			 * shall be wiped-out first, then, we update 'scan_done'
			 * to mark this depth will never generate results any more.
			 */
		}
		else
		{
			/* back to the previous depth to generate the source tuples. */
			if (WARP_READ_POS(wp,depth-1) + warpSize > WARP_WRITE_POS(wp,depth-1))
				return depth-1;
		}
	}

	read_pos = WARP_READ_POS(wp,depth-1) + LaneId();
	if (read_pos < WARP_WRITE_POS(wp,depth-1))
	{
		uint32_t	index = l_state++;

		read_pos = (read_pos % UNIT_TUPLES_PER_DEPTH);
		kcxt->kvars_slot = (kern_variable *)
			(src_kvars_addr_wp + read_pos * kcxt->kvars_nbytes);
		kcxt->kvars_class = (int *)(kcxt->kvars_slot + kcxt->kvars_nslots);
		if (index < kds_heap->nitems)
		{
			kern_tupitem *tupitem;
			uint32_t	offset = KDS_GET_ROWINDEX(kds_heap)[index];
			xpu_int4_t	status;

			tupitem = (kern_tupitem *)((char *)kds_heap +
									   kds_heap->length -
									   __kds_unpack(offset));
			kexp = SESSION_KEXP_JOIN_LOAD_VARS(kcxt->session, depth-1);
			ExecLoadVarsHeapTuple(kcxt, kexp, depth, kds_heap, &tupitem->htup);
			kexp = SESSION_KEXP_JOIN_QUALS(kcxt->session, depth-1);
			if (EXEC_KERN_EXPRESSION(kcxt, kexp, &status))
			{
				assert(!XPU_DATUM_ISNULL(&status));
				if (status.value > 0)
					tuple_is_valid = true;
				if (status.value != 0)
					matched = true;
			}
			if (oj_map && matched)
			{
				assert(tupitem->rowid < kds_heap->nitems);
				oj_map[tupitem->rowid] = true;
			}
		}
		else if (kmrels->chunks[depth-1].left_outer &&
				 index >= kds_heap->nitems && !matched)
		{
			/* fill up NULL fields, if FULL/LEFT OUTER JOIN */
			kexp = SESSION_KEXP_JOIN_LOAD_VARS(kcxt->session, depth-1);
			ExecLoadVarsHeapTuple(kcxt, kexp, depth, kds_heap, NULL);
			tuple_is_valid = true;
		}
	}
	else
	{
		l_state = UINT_MAX;
	}
	/* error checks */
	if (__any_sync(__activemask(), kcxt->errcode != ERRCODE_STROM_SUCCESS))
		return -1;
	/* save the result */
	mask = __ballot_sync(__activemask(), tuple_is_valid);
	if (LaneId() == 0)
	{
		write_pos = WARP_WRITE_POS(wp,depth);
		WARP_WRITE_POS(wp,depth) += __popc(mask);
	}
	write_pos = __shfl_sync(__activemask(), write_pos, 0);
	mask &= ((1U << LaneId()) - 1);
	write_pos += __popc(mask);

	if (tuple_is_valid)
	{
		write_pos = (write_pos % UNIT_TUPLES_PER_DEPTH);
		memcpy(dst_kvars_addr_wp + write_pos * kcxt->kvars_nbytes,
			   kcxt->kvars_slot,
			   kcxt->kvars_nbytes);
	}
	__syncwarp();
	if (WARP_WRITE_POS(wp,depth) >= WARP_READ_POS(wp,depth) + warpSize)
		return depth+1;
	return depth;
}

/*
 * GPU Hash-Join
 */
STATIC_FUNCTION(int)
execGpuJoinHashJoin(kern_context *kcxt,
					kern_warp_context *wp,
					kern_multirels *kmrels,
					int			depth,
					char	   *src_kvars_addr_wp,
					char	   *dst_kvars_addr_wp,
					uint32_t   &l_state,
					bool	   &matched)
{
	kern_data_store *kds_hash = KERN_MULTIRELS_INNER_KDS(kmrels, depth-1);
	bool	   *oj_map = KERN_MULTIRELS_OUTER_JOIN_MAP(kmrels, depth-1);
	kern_expression *kexp = NULL;
	kern_hashitem *khitem = NULL;
	uint32_t	read_pos;
	uint32_t	write_pos;
	uint32_t	index;
	uint32_t	mask;
	bool		tuple_is_valid = false;

	if (WARP_WRITE_POS(wp,depth) >= WARP_READ_POS(wp,depth) + warpSize)
	{
		/*
		 * Next depth already keeps warpSize or more pending tuples,
		 * so wipe out these tuples first.
		 */
		return depth+1;
	}

	if (__all_sync(__activemask(), l_state == UINT_MAX))
	{
		/*
		 * OK, all the threads in this warp reached to the end of hash-slot
		 * chain. Due to the above checks, the next depth has enough space
		 * to store the result in this depth.
		 * So, we process this depth again (if we have enough pending tuples),
		 * back to the previsou depth (if we don't have enough pending tuples
		 * in this depth), or move to the next depth if previous depth already
		 * reached to end of the chunk.
		 */
		if (LaneId() == 0)
			WARP_READ_POS(wp,depth-1) = Min(WARP_READ_POS(wp,depth-1) + warpSize,
											WARP_WRITE_POS(wp,depth-1));
		__syncwarp();
		l_state = 0;
		matched = false;
		if (wp->scan_done < depth)
		{
			/*
			 * The previous depth still may generate the source tuple.
			 */
			if (WARP_WRITE_POS(wp,depth-1) < WARP_READ_POS(wp,depth-1) + warpSize)
				return depth-1;
		}
		else
		{
			assert(wp->scan_done == depth);
			if (WARP_READ_POS(wp,depth-1) >= WARP_WRITE_POS(wp,depth-1))
			{
				if (LaneId() == 0)
					wp->scan_done = depth+1;
				return depth+1;
			}
			/*
			 * Elsewhere, remaining tuples in the combination buffer
			 * shall be wiped-out first, then, we update 'scan_done'
			 * to mark this depth will never generate results any more.
			 */
		}
	}
	write_pos = WARP_WRITE_POS(wp,depth-1);
	read_pos = WARP_READ_POS(wp,depth-1) + LaneId();
	index = (read_pos % UNIT_TUPLES_PER_DEPTH);
	kcxt->kvars_slot = (kern_variable *)
		(src_kvars_addr_wp + index * kcxt->kvars_nbytes);
	kcxt->kvars_class = (int *)(kcxt->kvars_slot + kcxt->kvars_nslots);

	if (l_state == 0)
	{
		/* pick up the first item from the hash-slot */
		if (read_pos < write_pos)
		{
			xpu_int4_t	hash;

			kexp = SESSION_KEXP_HASH_VALUE(kcxt->session, depth-1);
			if (EXEC_KERN_EXPRESSION(kcxt, kexp, &hash))
			{
				assert(!XPU_DATUM_ISNULL(&hash));
				for (khitem = KDS_HASH_FIRST_ITEM(kds_hash, hash.value);
					 khitem != NULL && khitem->hash != hash.value;
					 khitem = KDS_HASH_NEXT_ITEM(kds_hash, khitem->next));
			}
		}
		else
		{
			l_state = UINT_MAX;
		}
	}
	else if (l_state != UINT_MAX)
	{
		/* pick up the next one if any */
		uint32_t	hash_value;

		khitem = (kern_hashitem *)((char *)kds_hash + __kds_unpack(l_state));
		hash_value = khitem->hash;
		for (khitem = KDS_HASH_NEXT_ITEM(kds_hash, khitem->next);
			 khitem != NULL && khitem->hash != hash_value;
			 khitem = KDS_HASH_NEXT_ITEM(kds_hash, khitem->next));
	}
	/* error checks */
	if (__any_sync(__activemask(), kcxt->errcode != ERRCODE_STROM_SUCCESS))
		return -1;

	if (khitem)
	{
		xpu_int4_t	status;

		kexp = SESSION_KEXP_JOIN_LOAD_VARS(kcxt->session, depth-1);
		ExecLoadVarsHeapTuple(kcxt, kexp, depth, kds_hash, &khitem->t.htup);
		kexp = SESSION_KEXP_JOIN_QUALS(kcxt->session, depth-1);
		if (EXEC_KERN_EXPRESSION(kcxt, kexp, &status))
		{
			assert(!XPU_DATUM_ISNULL(&status));
			if (status.value > 0)
				tuple_is_valid = true;
			if (status.value != 0)
				matched = true;
		}
		if (oj_map && matched)
		{
			assert(khitem->t.rowid < kds_hash->nitems);
			oj_map[khitem->t.rowid] = true;
		}
		l_state = __kds_packed((char *)khitem - (char *)kds_hash);
	}
	else
	{
		if (kmrels->chunks[depth-1].left_outer &&
			l_state != UINT_MAX && !matched)
		{
			/* load NULL values on the inner portion */
			 kexp = SESSION_KEXP_JOIN_LOAD_VARS(kcxt->session, depth-1);
			 ExecLoadVarsHeapTuple(kcxt, kexp, depth, kds_hash, NULL);
			 tuple_is_valid = true;
		}
		l_state = UINT_MAX;
	}
	/* error checks */
	if (__any_sync(__activemask(), kcxt->errcode != ERRCODE_STROM_SUCCESS))
		return -1;
	/* save the result on the destination buffer */
	mask = __ballot_sync(__activemask(), tuple_is_valid);
	if (LaneId() == 0)
	{
		write_pos = WARP_WRITE_POS(wp,depth);
		WARP_WRITE_POS(wp,depth) += __popc(mask);
	}
	write_pos = __shfl_sync(__activemask(), write_pos, 0);
	mask &= ((1U << LaneId()) - 1);
	write_pos += __popc(mask);
	if (tuple_is_valid)
	{
		index = write_pos % UNIT_TUPLES_PER_DEPTH;
		memcpy(dst_kvars_addr_wp + index * kcxt->kvars_nbytes,
			   kcxt->kvars_slot,
			   kcxt->kvars_nbytes);
	}
	__syncwarp();
	if (WARP_WRITE_POS(wp,depth) >= WARP_READ_POS(wp,depth) + warpSize)
		return depth+1;
	return depth;
}

/*
 * gpujoin_prep_gistindex
 */
KERNEL_FUNCTION(void)
gpujoin_prep_gistindex(kern_multirels *kmrels, int depth)
{
	kern_data_store *kds_hash = KERN_MULTIRELS_INNER_KDS(kmrels, depth-1);
	kern_data_store *kds_gist = KERN_MULTIRELS_GIST_INDEX(kmrels, depth-1);
	BlockNumber		block_nr;
	OffsetNumber	i, maxoff;

	assert(kds_hash && kds_hash->format == KDS_FORMAT_HASH &&
		   kds_gist && kds_gist->format == KDS_FORMAT_BLOCK);
	for (block_nr = get_group_id();
		 block_nr < kds_gist->nitems;
		 block_nr += get_num_groups())
	{
		PageHeaderData *gist_page;
		ItemIdData	   *lpp;
		IndexTupleData *itup;
		kern_hashitem  *khitem;
		uint32_t		hash, t_off;

		gist_page = KDS_BLOCK_PGPAGE(kds_gist, block_nr);
		if (!GistPageIsLeaf(gist_page))
			continue;
		maxoff = PageGetMaxOffsetNumber(gist_page);
		for (i = get_local_id(); i < maxoff; i += get_local_size())
		{
			lpp = PageGetItemId(gist_page, i+1);
			if (ItemIdIsDead(lpp))
				continue;
			itup = (IndexTupleData *)PageGetItem(gist_page, lpp);

			/* lookup kds_hash */
			hash = pg_hash_any(&itup->t_tid, sizeof(ItemPointerData));
			for (khitem = KDS_HASH_FIRST_ITEM(kds_hash, hash);
				 khitem != NULL;
				 khitem = KDS_HASH_NEXT_ITEM(kds_hash, khitem->next))
			{
				if (ItemPointerEquals(&khitem->t.htup.t_ctid, &itup->t_tid))
				{
					t_off = __kds_packed((char *)&khitem->t.htup -
										 (char *)kds_hash);
					itup->t_tid.ip_blkid.bi_hi = (t_off >> 16);
					itup->t_tid.ip_blkid.bi_lo = (t_off & 0x0000ffffU);
					itup->t_tid.ip_posid = InvalidOffsetNumber;
					break;
				}
			}
			/* invalidate this leaf item, if not exist on kds_hash */
			if (!khitem)
				lpp->lp_flags = LP_DEAD;
		}
	}
}

/*
 * GiST-INDEX-JOIN
 */
STATIC_FUNCTION(int)
execGpuJoinGiSTJoin(kern_context *kcxt,
					kern_warp_context *wp,
					kern_multirels *kmrels,
					int         depth,
					char       *src_kvars_addr_wp,
					char       *dst_kvars_addr_wp,
					const kern_expression *kexp_gist,
					char	   *gist_kvars_addr_wp,
					uint32_t   &l_state,
					bool       &matched)
{
	kern_data_store *kds_hash = KERN_MULTIRELS_INNER_KDS(kmrels, depth-1);
	kern_data_store *kds_gist = KERN_MULTIRELS_GIST_INDEX(kmrels, depth-1);
	int				gist_depth = kexp_gist->u.gist.gist_depth;
	uint32_t		mask, index;
	uint32_t		read_pos;
	uint32_t		write_pos;

	assert(kds_hash && kds_hash->format == KDS_FORMAT_HASH &&
		   kds_gist && kds_gist->format == KDS_FORMAT_BLOCK);

	if (wp->scan_done > depth)
	{
		/*
		 * This depth will not generate any more tuples, so we move to
		 * the next level.
		 */
		return depth+1;
	}

	if (WARP_WRITE_POS(wp,depth) >= WARP_READ_POS(wp,depth) + warpSize)
	{
		/*
		 * Next depth already have warpSize or more pending tuples,
		 * so wipe out these tuples first.
		 */
		return depth+1;
	}

	if (WARP_WRITE_POS(wp,gist_depth)  >= WARP_READ_POS(wp,gist_depth) + warpSize ||
		(wp->scan_done >= depth &&		/* is terminal case? */
		 WARP_WRITE_POS(wp,depth-1) == WARP_READ_POS(wp,depth-1) &&
		 __all_sync(__activemask(), l_state == UINT_MAX)))
	{
		/*
		 * We already have 32 or more pending tuples; that is fetched by
		 * the GiST-index. So, try to fetch Join-Quals for these tuples.
		 */
		bool	join_is_valid = false;

		read_pos = WARP_READ_POS(wp,gist_depth) + LaneId();
		if (read_pos < WARP_WRITE_POS(wp,gist_depth))
		{
			const kern_expression *kexp_load
				= SESSION_KEXP_JOIN_LOAD_VARS(kcxt->session, depth-1);
			const kern_expression *kexp_join
				= SESSION_KEXP_JOIN_QUALS(kcxt->session, depth-1);

			index = (read_pos % UNIT_TUPLES_PER_DEPTH);
			kcxt->kvars_slot = (kern_variable *)
				(gist_kvars_addr_wp + index * kcxt->kvars_nbytes);
			kcxt->kvars_class = (int *)(kcxt->kvars_slot + kcxt->kvars_nslots);

			join_is_valid = ExecGiSTIndexPostQuals(kcxt, depth,
												   kds_hash,
												   kexp_gist,
												   kexp_load,
												   kexp_join);
		}
		/* error checks */
		if (__any_sync(__activemask(), kcxt->errcode != ERRCODE_STROM_SUCCESS))
			return -1;
		if (LaneId() == 0)
			WARP_READ_POS(wp,gist_depth) = Max(WARP_READ_POS(wp,gist_depth) + warpSize,
											   WARP_WRITE_POS(wp,gist_depth));

		mask = __ballot_sync(__activemask(), join_is_valid);
		if (LaneId() == 0)
		{
			write_pos = WARP_WRITE_POS(wp,depth);
			WARP_WRITE_POS(wp,depth) += __popc(mask);
		}
		write_pos = __shfl_sync(__activemask(), write_pos, 0);
		mask &= ((1U << LaneId()) - 1);
		write_pos += __popc(mask);
		if (join_is_valid)
		{
			index = write_pos % UNIT_TUPLES_PER_DEPTH;
			memcpy(dst_kvars_addr_wp + index * kcxt->kvars_nbytes,
				   kcxt->kvars_slot,
				   kcxt->kvars_nbytes);
		}
		__syncwarp();
		/* termination checks */
		if (LaneId() == 0 &&
			wp->scan_done >= depth &&
			WARP_WRITE_POS(wp,depth-1) == WARP_READ_POS(wp,depth-1) &&
			WARP_WRITE_POS(wp,gist_depth) <= WARP_READ_POS(wp,gist_depth) &&
			__all_sync(__activemask(), l_state == UINT_MAX))
		{
			assert(wp->scan_done == depth);
			wp->scan_done++;
			depth++;
		}
		return __shfl_sync(__activemask(), depth, 0);
	}

	if (__all_sync(__activemask(), l_state == UINT_MAX))
	{
		/*
		 * OK, all the threads in this warp reached to the end of the GiST
		 * index tree. Due to the above checks, the next depth has enough
		 * space to store the result in this depth.
		 */
		if (LaneId() == 0)
			WARP_READ_POS(wp,depth-1) = Min(WARP_READ_POS(wp,depth-1) + warpSize,
											WARP_WRITE_POS(wp,depth-1));
		__syncwarp();
		l_state = 0;
		matched = false;
		if (wp->scan_done < depth)
		{
			/* back to the previous depth; that still may generate source tuples */
			if (WARP_WRITE_POS(wp,depth-1) < WARP_READ_POS(wp,depth-1) + warpSize)
				return depth-1;
		}
		else
		{
			assert(wp->scan_done == depth);
			if (WARP_WRITE_POS(wp,depth-1) <= WARP_READ_POS(wp,depth-1))
			{
				/* wipe out the remaining tuples */
				return depth;
			}
			/*
			 * Elsewhere, the pending source tuples should be processed
			 * first, then, we update the 'scan_done' to mark this depth
			 * will never generate any results.
			 */
		}
	}

	/*
	 * Restart GiST-index scan from the head, or the previous position
	 */
	read_pos = WARP_READ_POS(wp,depth-1) + LaneId();
	if (read_pos < WARP_WRITE_POS(wp,depth-1))
	{
		if (l_state != UINT_MAX)
		{
			index = (read_pos % UNIT_TUPLES_PER_DEPTH);
			kcxt->kvars_slot = (kern_variable *)
				(src_kvars_addr_wp + index * kcxt->kvars_nbytes);
			kcxt->kvars_class = (int *)(kcxt->kvars_slot + kcxt->kvars_nslots);

			l_state = ExecGiSTIndexGetNext(kcxt,
										   kds_hash,
										   kds_gist,
										   kexp_gist,
										   l_state);
		}
	}
	else
	{
		l_state = UINT_MAX;
	}
	/* error checks */
	if (__any_sync(__activemask(), kcxt->errcode != ERRCODE_STROM_SUCCESS))
		return -1;
	/* save the result on the destination buffer */
	mask = __ballot_sync(__activemask(), l_state != UINT_MAX);
	if (LaneId() == 0)
	{
		write_pos = WARP_WRITE_POS(wp,gist_depth);
		WARP_WRITE_POS(wp,gist_depth) += __popc(mask);
	}
	write_pos = __shfl_sync(__activemask(), write_pos, 0);
	mask &= ((1U << LaneId()) - 1);
	write_pos += __popc(mask);
	if (l_state != UINT_MAX)
	{
		index = write_pos % UNIT_TUPLES_PER_DEPTH;

		memcpy(gist_kvars_addr_wp + index * kcxt->kvars_nbytes,
			   kcxt->kvars_slot,
			   kcxt->kvars_nbytes);
	}
	__syncwarp();
	return depth;
}

/*
 * GPU Projection
 */
PUBLIC_FUNCTION(int)
execGpuJoinProjection(kern_context *kcxt,
					  kern_warp_context *wp,
					  int n_rels,	/* index of read/write-pos */
					  kern_data_store *kds_dst,
					  kern_expression *kexp_projection,
					  char *kvars_addr_wp,
					  bool *p_try_suspend)
{
	uint32_t	write_pos = WARP_WRITE_POS(wp,n_rels);
	uint32_t	read_pos = WARP_READ_POS(wp,n_rels);
	uint32_t	count;
	uint32_t	mask;
	uint32_t	row_id;
	uint32_t	offset;
	int			tupsz = 0;
	int			total_sz = 0;
	bool		try_suspend = false;
	union {
		struct {
			uint32_t	nitems;
			uint32_t	usage;
		} i;
		uint64_t		v64;
	} oldval, curval, newval;

	/*
	 * The previous depth still may produce new tuples, and number of
	 * the current result tuples is not sufficient to run projection.
	 */
	if (wp->scan_done <= n_rels && read_pos + warpSize > write_pos)
		return n_rels;

	read_pos += LaneId();
	if (read_pos < write_pos)
	{
		int			index = (read_pos % UNIT_TUPLES_PER_DEPTH);

		kcxt->kvars_slot = (kern_variable *)
			(kvars_addr_wp + index * kcxt->kvars_nbytes);
		kcxt->kvars_class = (int *)(kcxt->kvars_slot + kcxt->kvars_nslots);
		tupsz = kern_estimate_heaptuple(kcxt,
										kexp_projection,
										kds_dst);
		if (tupsz < 0)
			STROM_ELOG(kcxt, "unable to compute tuple size");
	}
	/* error checks */
	if (__any_sync(__activemask(), kcxt->errcode != ERRCODE_STROM_SUCCESS))
		return -1;
	/* allocation of the destination buffer */
	assert(kds_dst->format == KDS_FORMAT_ROW);
	mask = __ballot_sync(__activemask(), tupsz > 0);
	count = __popc(mask);
	mask &= ((1U << LaneId()) - 1);
	row_id = __popc(mask);
	assert(tupsz == 0 || row_id < count);

	offset = __reduce_stair_add_sync(tupsz, &total_sz);
	if (LaneId() == 0)
	{
		curval.i.nitems = kds_dst->nitems;
		curval.i.usage  = kds_dst->usage;
		do {
			newval = oldval = curval;
			newval.i.nitems += count;
			newval.i.usage  += __kds_packed(total_sz);

			if (KDS_HEAD_LENGTH(kds_dst) +
				MAXALIGN(sizeof(uint32_t) * newval.i.nitems) +
				__kds_unpack(newval.i.usage) > kds_dst->length)
			{
				try_suspend = true;
				break;
			}
		} while ((curval.v64 = atomicCAS((unsigned long long *)&kds_dst->nitems,
										 oldval.v64,
										 newval.v64)) != oldval.v64);
	}
	oldval.v64 = __shfl_sync(__activemask(), oldval.v64, 0);
	row_id += oldval.i.nitems;
	/* data store has no space? */
	if (__any_sync(__activemask(), try_suspend))
	{
		*p_try_suspend = true;
		return -1;
	}
	/* write out the tuple */
	if (tupsz > 0)
	{
		kern_tupitem   *tupitem;

		offset += __kds_unpack(oldval.i.usage);
		KDS_GET_ROWINDEX(kds_dst)[row_id] = __kds_packed(offset);
		tupitem = (kern_tupitem *)
			((char *)kds_dst + kds_dst->length - offset);
		tupitem->rowid = row_id;
		tupitem->t_len = kern_form_heaptuple(kcxt,
											 kexp_projection,
											 kds_dst,
											 &tupitem->htup);
	}
	/* update the read position */
	if (LaneId() == 0)
	{
		WARP_READ_POS(wp,n_rels) += count;
		assert(WARP_WRITE_POS(wp,n_rels) >= WARP_READ_POS(wp,n_rels));
	}
	__syncwarp();
	if (wp->scan_done <= n_rels)
	{
		if (WARP_WRITE_POS(wp,n_rels) < WARP_READ_POS(wp,n_rels) + warpSize)
			return n_rels;	/* back to the previous depth */
	}
	else
	{
		if (WARP_READ_POS(wp,n_rels) >= WARP_WRITE_POS(wp,n_rels))
			return -1;		/* ok, end of GpuJoin */
	}
	return n_rels + 1;		/* elsewhere, try again? */
}

/*
 * kern_gpujoin_main
 */
KERNEL_FUNCTION(void)
kern_gpujoin_main(kern_session_info *session,
				  kern_gputask *kgtask,
				  kern_multirels *kmrels,
				  kern_data_store *kds_src,
				  kern_data_extra *kds_extra,
				  kern_data_store *kds_dst)
{
	kern_context	   *kcxt;
	kern_warp_context  *wp, *wp_saved;
	char			   *kvars_addr_wp;
	uint32_t			kvars_chunksz;
	uint32_t		   *l_state;
	bool			   *matched;
	uint32_t			wp_base_sz;
	uint32_t			n_rels = (kmrels ? kmrels->num_rels : 0);
	int					depth;
	__shared__ uint32_t smx_row_count;

	assert(kgtask->kvars_nslots == session->kcxt_kvars_nslots &&
		   kgtask->kvars_nbytes == session->kcxt_kvars_nbytes &&
		   kgtask->kvars_ndims >= n_rels &&
		   kgtask->n_rels == n_rels);
	/* setup execution context */
	INIT_KERNEL_CONTEXT(kcxt, session);
	wp_base_sz = __KERN_WARP_CONTEXT_BASESZ(kgtask->kvars_ndims);
	wp = (kern_warp_context *)SHARED_WORKMEM(wp_base_sz, get_local_id() / warpSize);
	wp_saved = KERN_GPUTASK_WARP_CONTEXT(kgtask);
	l_state = KERN_GPUTASK_LSTATE_ARRAY(kgtask);
	matched = KERN_GPUTASK_MATCHED_ARRAY(kgtask);
	kvars_chunksz = kcxt->kvars_nbytes * UNIT_TUPLES_PER_DEPTH;
	kvars_addr_wp = (char *)wp_saved + wp_base_sz;

	if (kgtask->resume_context)
	{
		/* resume the warp-context from the previous execution */
		if (LaneId() == 0)
			memcpy(wp, wp_saved, wp_base_sz);
		if (get_local_id() == 0)
			smx_row_count = wp->smx_row_count;
		depth = __shfl_sync(__activemask(), wp->depth, 0);
	}
	else
	{
		/* zero clear the wp */
		if (LaneId() == 0)
			memset(wp, 0, wp_base_sz);
		if (get_local_id() == 0)
			smx_row_count = 0;
		depth = 0;
		if (l_state)
			memset(l_state, 0, sizeof(uint32_t) * kgtask->kvars_ndims);
		if (matched)
			memset(matched, 0, sizeof(bool)     * kgtask->kvars_ndims);
	}
	__syncthreads();

	/* main logic of GpuJoin */
	while (depth >= 0)
	{
		kcxt_reset(kcxt);
		if (depth == 0)
		{
			/* LOAD FROM THE SOURCE */
			depth = execGpuScanLoadSource(kcxt, wp,
										  kds_src,
										  kds_extra,
										  SESSION_KEXP_SCAN_LOAD_VARS(session),
										  SESSION_KEXP_SCAN_QUALS(session),
										  kvars_addr_wp,	/* depth=0 */
										  &smx_row_count);
		}
		else if (depth > n_rels)
		{
			bool	try_suspend = false;

			assert(depth == n_rels+1);
			if (session->xpucode_projection)
			{
				/* PROJECTION */
				depth = execGpuJoinProjection(kcxt, wp,
											  n_rels,
											  kds_dst,
											  SESSION_KEXP_PROJECTION(session),
											  kvars_addr_wp + kvars_chunksz * n_rels,
											  &try_suspend);
			}
			else
			{
				/* PRE-AGG */
				depth = execGpuPreAggGroupBy(kcxt, wp,
											 n_rels,
											 kds_dst,
											 kvars_addr_wp + kvars_chunksz * n_rels,
											 &try_suspend);
			}
			if (__any_sync(__activemask(), try_suspend))
			{
				if (LaneId() == 0)
					atomicAdd(&kgtask->suspend_count, 1);
				assert(depth < 0);
			}
		}
		else if (kmrels->chunks[depth-1].is_nestloop)
		{
			/* NEST-LOOP */
			depth = execGpuJoinNestLoop(kcxt, wp,
										kmrels,
										depth,
										kvars_addr_wp + kvars_chunksz * (depth-1),
										kvars_addr_wp + kvars_chunksz * depth,
										l_state[depth-1],	/* call by reference */
										matched[depth-1]);	/* call by reference */
		}
		else if (kmrels->chunks[depth-1].gist_offset != 0)
		{
			/* GiST-INDEX-JOIN */
			const kern_expression *kexp_gist
				= SESSION_KEXP_GIST_EVALS(kcxt->session, depth-1);

			assert(kexp_gist != NULL &&
				   kexp_gist->opcode == FuncOpCode__GiSTEval &&
				   kexp_gist->u.gist.gist_depth < kgtask->kvars_ndims);
			depth = execGpuJoinGiSTJoin(kcxt, wp,
										kmrels,
										depth,
										kvars_addr_wp + kvars_chunksz * (depth-1),
										kvars_addr_wp + kvars_chunksz * depth,
										kexp_gist,
										kvars_addr_wp + kvars_chunksz * kexp_gist->u.gist.gist_depth,
										l_state[depth-1],	/* call by reference */
										matched[depth-1]);	/* call by reference */
		}
		else
		{
			/* HASH-JOIN */
			depth = execGpuJoinHashJoin(kcxt, wp,
										kmrels,
										depth,
										kvars_addr_wp + kvars_chunksz * (depth-1),
										kvars_addr_wp + kvars_chunksz * depth,
										l_state[depth-1],	/* call by reference */
										matched[depth-1]);	/* call by reference */
		}
		assert(__shfl_sync(__activemask(), depth, 0) == depth);
		/* bailout if any error status */
		if (__any_sync(__activemask(), kcxt->errcode != ERRCODE_STROM_SUCCESS))
			break;
	}
	__syncthreads();

	if (LaneId() == 0)
	{
		/* update the statistics */
		if (depth < 0 && WARP_READ_POS(wp,n_rels) >= WARP_WRITE_POS(wp,n_rels))
		{
			/* number of raw-tuples fetched from the heap block */
			atomicAdd(&kgtask->nitems_raw, wp->lp_wr_pos);
			atomicAdd(&kgtask->nitems_in, WARP_WRITE_POS(wp, 0));
			for (int i=0; i < n_rels; i++)
			{
				const kern_expression *kexp_gist
					= SESSION_KEXP_GIST_EVALS(session, i);
				if (kexp_gist)
				{
					int		gist_depth = kexp_gist->u.gist.gist_depth;

					assert(gist_depth > n_rels &&
						   gist_depth < kgtask->kvars_ndims);
					atomicAdd(&kgtask->stats[i].nitems_gist,
							  WARP_WRITE_POS(wp, gist_depth));
				}
				atomicAdd(&kgtask->stats[i].nitems_out,
						  WARP_WRITE_POS(wp,i+1));
			}
			atomicAdd(&kgtask->nitems_out, WARP_WRITE_POS(wp, n_rels));
		}
		/* suspend the execution context */
		wp->depth = depth;
		wp->smx_row_count = smx_row_count;
		memcpy(wp_saved, wp, wp_base_sz);
	}
	STROM_WRITEBACK_ERROR_STATUS(&kgtask->kerror, kcxt);
}
