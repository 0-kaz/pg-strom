#include "hip/hip_runtime.h"
/*
 * xpu_numeric.cu
 *
 * collection of numeric type support for both of GPU and DPU
 * ----
 * Copyright 2011-2023 (C) KaiGai Kohei <kaigai@kaigai.gr.jp>
 * Copyright 2014-2023 (C) PG-Strom Developers Team
 *
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the PostgreSQL License.
 */
#include "xpu_common.h"
#include <math.h>

INLINE_FUNCTION(int)
xpu_numeric_sign(xpu_numeric_t *num)
{
	if (num->kind != XPU_NUMERIC_KIND__VALID)
	{
		Assert(num->kind != XPU_NUMERIC_KIND__NAN);
		/* Must be +Inf or -Inf */
		return num->kind == XPU_NUMERIC_KIND__POS_INF ? INT_MAX : INT_MIN;
	}
	if (num->value > 0)
		return 1;
	if (num->value < 0)
		return -1;
	return 0;
}

STATIC_FUNCTION(bool)
xpu_numeric_datum_heap_read(kern_context *kcxt,
							const void *addr,
							xpu_datum_t *__result)
{
	xpu_numeric_t  *result = (xpu_numeric_t *)__result;
	const char	   *errmsg;

	errmsg = __xpu_numeric_from_varlena(result, (const varlena *)addr);
	if (!errmsg)
		return true;
	result->expr_ops = NULL;
	STROM_ELOG(kcxt, errmsg);
	return false;
}

STATIC_FUNCTION(bool)
xpu_numeric_datum_arrow_read(kern_context *kcxt,
							 const kern_data_store *kds,
							 const kern_colmeta *cmeta,
							 uint32_t kds_index,
							 xpu_datum_t *__result)
{
	xpu_numeric_t  *result = (xpu_numeric_t *)__result;
	const int128_t *addr;

	if (cmeta->attopts.tag != ArrowType__Decimal)
	{
		STROM_ELOG(kcxt, "xpu_numeric_t must be mapped on Arrow::Decimal");
		return false;
	}
	if (cmeta->attopts.decimal.bitWidth != 128)
	{
		STROM_ELOG(kcxt, "Arrow::Decimal unsupported bitWidth");
		return false;
	}
	addr = (const int128_t *)KDS_ARROW_REF_SIMPLE_DATUM(kds, cmeta,
														kds_index,
														sizeof(int128_t));
	if (addr)
		set_normalized_numeric(result, *addr,
							   cmeta->attopts.decimal.scale);
	else
		result->expr_ops = NULL;
	return true;
}

STATIC_FUNCTION(bool)
xpu_numeric_datum_kvec_load(kern_context *kcxt,
							const kvec_datum_t *__kvecs,
							uint32_t kvecs_id,
							xpu_datum_t *__result)
{
	const kvec_numeric_t *kvecs = (const kvec_numeric_t *)__kvecs;
	xpu_numeric_t *result = (xpu_numeric_t *)__result;

	result->expr_ops = &xpu_numeric_ops;
	result->kind     = kvecs->kinds[kvecs_id];
	result->weight   = kvecs->weights[kvecs_id];
	result->value    = kvecs->values[kvecs_id];

    return true;
}

STATIC_FUNCTION(bool)
xpu_numeric_datum_kvec_save(kern_context *kcxt,
							const xpu_datum_t *__xdatum,
							kvec_datum_t *__kvecs,
							uint32_t kvecs_id)
{
	const xpu_numeric_t *xdatum = (const xpu_numeric_t *)__xdatum;
	kvec_numeric_t *kvecs = (kvec_numeric_t *)__kvecs;

	kvecs->kinds[kvecs_id]   = xdatum->kind;
	kvecs->weights[kvecs_id] = xdatum->weight;
	kvecs->values[kvecs_id]  = xdatum->value;
    return true;
}

STATIC_FUNCTION(bool)
xpu_numeric_datum_kvec_copy(kern_context *kcxt,
							const kvec_datum_t *__kvecs_src,
							uint32_t kvecs_src_id,
							kvec_datum_t *__kvecs_dst,
							uint32_t kvecs_dst_id)
{
	const kvec_numeric_t *kvecs_src = (const kvec_numeric_t *)__kvecs_src;
	kvec_numeric_t *kvecs_dst = (kvec_numeric_t *)__kvecs_dst;

	kvecs_dst->kinds[kvecs_dst_id]   = kvecs_src->kinds[kvecs_src_id];
	kvecs_dst->weights[kvecs_dst_id] = kvecs_src->weights[kvecs_src_id];
	kvecs_dst->values[kvecs_dst_id]  = kvecs_src->values[kvecs_src_id];
	return true;
}

STATIC_FUNCTION(int)
xpu_numeric_datum_write(kern_context *kcxt,
						char *buffer,
						const kern_colmeta *cmeta,
						const xpu_datum_t *__arg)
{
	const xpu_numeric_t *arg = (const xpu_numeric_t *)__arg;

	if (arg->kind != XPU_NUMERIC_KIND__VALID)
	{
		int		sz = offsetof(NumericData, choice.n_header) + sizeof(uint16_t);

		if (buffer)
		{
			NumericChoice *nc = (NumericChoice *)buffer;

			if (arg->kind == XPU_NUMERIC_KIND__POS_INF)
				nc->n_header = NUMERIC_PINF;
			else if (arg->kind == XPU_NUMERIC_KIND__NEG_INF)
				nc->n_header = NUMERIC_NINF;
			else
				nc->n_header = NUMERIC_NAN;
			SET_VARSIZE(nc, sz);
		}
		return sz;
	}
	return __xpu_numeric_to_varlena(buffer, arg->weight, arg->value);
}

PUBLIC_FUNCTION(bool)
xpu_numeric_datum_hash(kern_context *kcxt,
					   uint32_t *p_hash,
					   const xpu_datum_t *__arg)
{
	const xpu_numeric_t *arg = (const xpu_numeric_t *)__arg;

	if (XPU_DATUM_ISNULL(arg))
		*p_hash = 0;
	else if (arg->kind != XPU_NUMERIC_KIND__VALID)
		*p_hash = pg_hash_any(&arg->kind, sizeof(uint8_t));
	else
		*p_hash = (pg_hash_any(&arg->weight, sizeof(int16_t)) ^
				   pg_hash_any(&arg->value, sizeof(int128_t)));
	return true;
}

STATIC_FUNCTION(int)
__numeric_compare(const xpu_numeric_t *a, const xpu_numeric_t *b);

STATIC_FUNCTION(bool)
xpu_numeric_datum_comp(kern_context *kcxt,
					   int *p_comp,
					   const xpu_datum_t *__a,
					   const xpu_datum_t *__b)
{
	const xpu_numeric_t *a = (const xpu_numeric_t *)__a;
	const xpu_numeric_t *b = (const xpu_numeric_t *)__b;

	assert(!XPU_DATUM_ISNULL(a) && !XPU_DATUM_ISNULL(b));
	*p_comp = __numeric_compare(a, b);
	return true;
}
PGSTROM_SQLTYPE_OPERATORS(numeric, false, 4, -1);

PUBLIC_FUNCTION(bool)
__xpu_numeric_to_int64(kern_context *kcxt,
					   int64_t *p_ival,
					   const xpu_numeric_t *num,
					   int64_t min_value,
					   int64_t max_value)
{
	int128_t	ival;
	int16_t		weight;

	assert(num->expr_ops == &xpu_numeric_ops);

	if (num->kind != XPU_NUMERIC_KIND__VALID)
	{
		STROM_ELOG(kcxt, "cannot convert NaN/Inf to integer");
		return false;
	}

	ival = num->value;
	weight = num->weight;
	if (ival != 0)
	{
		while (weight > 0)
		{
			/* round of 0.x digit */
			if (weight == 1)
				ival += (ival > 0 ? 5 : -5);
			ival /= 10;
			weight--;
		}
		while (weight < 0)
		{
			ival *= 10;
			weight++;
			if (ival < min_value || ival > max_value)
				break;
		}
		if (ival < min_value || ival > max_value)
		{
			STROM_ELOG(kcxt, "integer out of range");
			return false;
		}
	}
	*p_ival = ival;
	return true;
}

#define PG_NUMERIC_TO_INT_TEMPLATE(TARGET,MIN_VALUE,MAX_VALUE)		\
	PUBLIC_FUNCTION(bool)											\
	pgfn_numeric_to_##TARGET(XPU_PGFUNCTION_ARGS)					\
	{																\
		int64_t		ival;											\
		KEXP_PROCESS_ARGS1(TARGET, numeric, num);					\
																	\
		if (XPU_DATUM_ISNULL(&num))									\
		{															\
			result->expr_ops = NULL;								\
		}															\
		else if (!__xpu_numeric_to_int64(kcxt, &ival, &num,			\
										 MIN_VALUE, MAX_VALUE))		\
		{															\
			return false;											\
		}															\
		else														\
		{															\
			result->value = ival;									\
			result->expr_ops = &xpu_##TARGET##_ops;					\
		}															\
		return true;												\
	}
PG_NUMERIC_TO_INT_TEMPLATE(int1,SCHAR_MIN,SCHAR_MAX)
PG_NUMERIC_TO_INT_TEMPLATE(int2,SHRT_MIN,SHRT_MAX)
PG_NUMERIC_TO_INT_TEMPLATE(int4,INT_MIN,INT_MAX)
PG_NUMERIC_TO_INT_TEMPLATE(int8,LLONG_MIN,LLONG_MAX)

PUBLIC_FUNCTION(bool)
pgfn_numeric_to_money(XPU_PGFUNCTION_ARGS)
{
	int64_t		ival;
	KEXP_PROCESS_ARGS1(money, numeric, num);

	if (XPU_DATUM_ISNULL(&num))
		result->expr_ops = NULL;
	else
	{
		const kern_session_info *session = kcxt->session;
		int		fpoint = session->session_currency_frac_digits;

		if (fpoint < 0 || fpoint > 10)
			fpoint = 2;
		num.weight -= fpoint;
		if (!__xpu_numeric_to_int64(kcxt, &ival, &num,
									LLONG_MIN,LLONG_MAX))
			return false;
		result->expr_ops = &xpu_money_ops;
		result->value = ival;
	}
	return true;
}

PUBLIC_FUNCTION(bool)
__xpu_numeric_to_fp64(kern_context *kcxt,
					  float8_t *p_fval,
					  const xpu_numeric_t *num)
{
	if (num->kind == XPU_NUMERIC_KIND__VALID)
	{
		float8_t	fval = num->value;
		int16_t		weight = num->weight;

		if (fval != 0.0)
		{
			while (weight > 0)
			{
				fval /= 10.0;
				weight--;
			}
			while (weight < 0)
			{
				fval *= 10.0;
				weight++;
			}
			if (isnan(fval) || isinf(fval))
			{
				STROM_ELOG(kcxt,"float out of range");
				return false;
			}
		}
		*p_fval = fval;
	}
	else if (num->kind == XPU_NUMERIC_KIND__POS_INF)
		*p_fval = INFINITY;
	else if (num->kind == XPU_NUMERIC_KIND__NEG_INF)
		*p_fval = -INFINITY;
	else
		*p_fval = NAN;

	return true;
}

#define PG_NUMERIC_TO_FLOAT_TEMPLATE(TARGET,__CAST)					\
	PUBLIC_FUNCTION(bool)											\
	pgfn_numeric_to_##TARGET(XPU_PGFUNCTION_ARGS)					\
	{																\
		float8_t		fval;										\
		KEXP_PROCESS_ARGS1(TARGET, numeric, num);					\
																	\
		if (XPU_DATUM_ISNULL(&num))									\
			result->expr_ops = NULL;								\
		else if (!__xpu_numeric_to_fp64(kcxt, &fval, &num))			\
			return false;											\
		else														\
		{															\
			result->expr_ops = &xpu_##TARGET##_ops;					\
			result->value = __CAST(fval);							\
		}															\
		return true;												\
	}
PG_NUMERIC_TO_FLOAT_TEMPLATE(float2, __to_fp16)
PG_NUMERIC_TO_FLOAT_TEMPLATE(float4, __to_fp32)
PG_NUMERIC_TO_FLOAT_TEMPLATE(float8, __to_fp64)

#define PG_INT_TO_NUMERIC_TEMPLATE(SOURCE)							\
	PUBLIC_FUNCTION(bool)											\
	pgfn_##SOURCE##_to_numeric(XPU_PGFUNCTION_ARGS)					\
	{																\
		KEXP_PROCESS_ARGS1(numeric, SOURCE, ival);					\
																	\
		if (XPU_DATUM_ISNULL(&ival))								\
			result->expr_ops = NULL;								\
		else														\
		{															\
			result->expr_ops = &xpu_numeric_ops;					\
			set_normalized_numeric(result, ival.value, 0);			\
		}															\
		return true;												\
	}
PG_INT_TO_NUMERIC_TEMPLATE(int1)
PG_INT_TO_NUMERIC_TEMPLATE(int2)
PG_INT_TO_NUMERIC_TEMPLATE(int4)
PG_INT_TO_NUMERIC_TEMPLATE(int8)

PUBLIC_FUNCTION(bool)
pgfn_money_to_numeric(XPU_PGFUNCTION_ARGS)
{
	KEXP_PROCESS_ARGS1(numeric, money, ival);
	if (XPU_DATUM_ISNULL(&ival))
		result->expr_ops = NULL;
	else
	{
		const kern_session_info *session = kcxt->session;
		int		fpoint = session->session_currency_frac_digits;

		if (fpoint < 0 || fpoint > 10)
			fpoint = 2;

		result->expr_ops = &xpu_numeric_ops;
		set_normalized_numeric(result, ival.value, 0);
		result->weight += fpoint;
	}
	return true;
}

#define PG_FLOAT_TO_NUMERIC_TEMPLATE(SOURCE,__TYPE,__CAST,			\
									 __MODF,__RINTL)				\
	PUBLIC_FUNCTION(bool)											\
	pgfn_##SOURCE##_to_numeric(XPU_PGFUNCTION_ARGS)					\
	{																\
		xpu_numeric_t	   *result = (xpu_numeric_t *)__result;		\
		xpu_##SOURCE##_t	datum;									\
		__TYPE				fval;									\
		const kern_expression *karg = KEXP_FIRST_ARG(kexp);			\
																	\
		assert(kexp->nr_args == 1 &&								\
			   KEXP_IS_VALID(karg,SOURCE));							\
		if (!EXEC_KERN_EXPRESSION(kcxt, karg, &datum))				\
			return false;											\
		if (XPU_DATUM_ISNULL(&datum))								\
			result->expr_ops = NULL;								\
		else														\
		{															\
			result->expr_ops = &xpu_numeric_ops;					\
			fval = __CAST(datum.value);								\
			if (isinf(fval))										\
				result->kind = (fval > 0.0							\
								? XPU_NUMERIC_KIND__POS_INF			\
								: XPU_NUMERIC_KIND__NEG_INF);		\
			else if (isnan(fval))									\
				result->kind = XPU_NUMERIC_KIND__NAN;				\
			else													\
			{														\
				__TYPE		a,b = __MODF(fval, &a);					\
				int128_t	value = (int128_t)a;					\
				int16_t		weight = 0;								\
				bool		negative = (value < 0);					\
																	\
				if (negative)										\
					value = -value;									\
				while (b != 0.0 && (value>>124) == 0)				\
				{													\
					b = __MODF(b * 10.0, &a);						\
					value = 10 * value + (int128_t)a;				\
					weight++;										\
				}													\
				set_normalized_numeric(result,value,weight);		\
			}														\
		}															\
		return true;												\
	}
PG_FLOAT_TO_NUMERIC_TEMPLATE(float2, float,__to_fp32,modff,rintf)
PG_FLOAT_TO_NUMERIC_TEMPLATE(float4, float,__to_fp32,modff,rintf)
PG_FLOAT_TO_NUMERIC_TEMPLATE(float8,double,__to_fp64,modf, rint)

STATIC_FUNCTION(int)
__numeric_compare(const xpu_numeric_t *a, const xpu_numeric_t *b)
{
	int128_t	a_val = a->value;
	int128_t	b_val = b->value;
	int16_t		a_weight = a->weight;
	int16_t		b_weight = b->weight;

	/* If any NaN or Inf */
	if (a->kind != XPU_NUMERIC_KIND__VALID)
	{
		if (a->kind == XPU_NUMERIC_KIND__NAN)
		{
			if (b->kind == XPU_NUMERIC_KIND__NAN)
				return 0;	/* NaN == Nan */
			return 1;		/* NaN > non-NaN */
		}
		else if (a->kind == XPU_NUMERIC_KIND__POS_INF)
		{
			if (b->kind == XPU_NUMERIC_KIND__NAN)
				return -1;	/* +Inf < NaN */
			if (b->kind == XPU_NUMERIC_KIND__POS_INF)
				return 0;	/* +Inf == +Inf */
			return 1;		/* +Inf > anything else */
		}
		else
		{
			if (b->kind == XPU_NUMERIC_KIND__NEG_INF)
				return 0;	/* -Inf == -Inf */
			return -1;		/* -Inf < anything else */
		}
	}
	else if (b->kind != XPU_NUMERIC_KIND__VALID)
	{
		if (b->kind == XPU_NUMERIC_KIND__NEG_INF)
			return 1;		/* normal > -Inf */
		else
			return -1;		/* normal < NaN or +Inf */
	}
	else if ((a_val > 0 && b_val <= 0) || (a_val == 0 && b_val < 0))
		return 1;
	else if ((b_val > 0 && a_val <= 0) || (b_val == 0 && a_val < 0))
		return -1;
	/* Ok, both side are same sign with valid values */
	while (a_weight > b_weight)
	{
		b_val *= 10;
		b_weight++;
	}
	while (a_weight < b_weight)
	{
		a_val *= 10;
		a_weight++;
	}
	if (a_val > b_val)
		return 1;
	if (a_val < b_val)
		return -1;
	return 0;
}

#define PG_NUMERIC_COMPARE_TEMPLATE(NAME,OPER)							\
	PUBLIC_FUNCTION(bool)												\
	pgfn_numeric_##NAME(XPU_PGFUNCTION_ARGS)							\
	{																	\
		xpu_bool_t	   *result = (xpu_bool_t *)__result;				\
		xpu_numeric_t	datum_a, datum_b;								\
		const kern_expression *karg = KEXP_FIRST_ARG(kexp);				\
																		\
		assert(kexp->nr_args == 2 &&									\
			   KEXP_IS_VALID(karg,numeric));							\
		if (!EXEC_KERN_EXPRESSION(kcxt, karg, &datum_a))				\
			return false;												\
		karg = KEXP_NEXT_ARG(karg);										\
		assert(KEXP_IS_VALID(karg, numeric));							\
		if (!EXEC_KERN_EXPRESSION(kcxt, karg, &datum_b))				\
			return false;												\
		if (XPU_DATUM_ISNULL(&datum_a) || XPU_DATUM_ISNULL(&datum_b))	\
		{																\
			__pg_simple_nullcomp_##NAME(&datum_a, &datum_b);			\
		}																\
		else															\
		{																\
			result->expr_ops = &xpu_numeric_ops;						\
			result->value = (__numeric_compare(&datum_a,				\
											   &datum_b) OPER 0);		\
		}																\
		return true;													\
	}
PG_NUMERIC_COMPARE_TEMPLATE(eq, ==)
PG_NUMERIC_COMPARE_TEMPLATE(ne, !=)
PG_NUMERIC_COMPARE_TEMPLATE(lt, <)
PG_NUMERIC_COMPARE_TEMPLATE(le, <=)
PG_NUMERIC_COMPARE_TEMPLATE(gt, >)
PG_NUMERIC_COMPARE_TEMPLATE(ge, >=)

PUBLIC_FUNCTION(bool)
pgfn_numeric_add(XPU_PGFUNCTION_ARGS)
{
	xpu_numeric_t  *result = (xpu_numeric_t *)__result;
	xpu_numeric_t	datum_a;
	xpu_numeric_t	datum_b;
	const kern_expression *karg = KEXP_FIRST_ARG(kexp);

	assert(kexp->nr_args == 2 &&
		   KEXP_IS_VALID(karg, numeric));
	if (!EXEC_KERN_EXPRESSION(kcxt, karg, &datum_a))
		return false;
	karg = KEXP_NEXT_ARG(karg);
	assert(KEXP_IS_VALID(karg, numeric));
	if (!EXEC_KERN_EXPRESSION(kcxt, karg, &datum_b))
		return false;
	if (XPU_DATUM_ISNULL(&datum_a) || XPU_DATUM_ISNULL(&datum_b))
		result->expr_ops = NULL;
	else
	{
		result->expr_ops = &xpu_numeric_ops;

		if (datum_a.kind != XPU_NUMERIC_KIND__VALID ||
			datum_b.kind != XPU_NUMERIC_KIND__VALID)
		{
			if (datum_a.kind == XPU_NUMERIC_KIND__NAN ||
				datum_b.kind == XPU_NUMERIC_KIND__NAN)
				result->kind = XPU_NUMERIC_KIND__NAN;
			else if (datum_a.kind == XPU_NUMERIC_KIND__POS_INF)
			{
				if (datum_b.kind == XPU_NUMERIC_KIND__NEG_INF)
					result->kind = XPU_NUMERIC_KIND__NAN;	/* Inf - Inf */
				else
					result->kind = XPU_NUMERIC_KIND__POS_INF;
			}
			else if (datum_a.kind == XPU_NUMERIC_KIND__NEG_INF)
			{
				if (datum_b.kind == XPU_NUMERIC_KIND__POS_INF)
					result->kind = XPU_NUMERIC_KIND__NAN;	/* -Inf + Inf */
				else
					result->kind = XPU_NUMERIC_KIND__NEG_INF;
			}
			else if (datum_b.kind == XPU_NUMERIC_KIND__POS_INF)
				result->kind = XPU_NUMERIC_KIND__POS_INF;
			else
				result->kind = XPU_NUMERIC_KIND__NEG_INF;
		}
		else
		{
			while (datum_a.weight > datum_b.weight)
			{
				datum_b.value *= 10;
				datum_b.weight++;
			}
			while (datum_a.weight < datum_b.weight)
			{
				datum_a.value *= 10;
				datum_a.weight++;
			}
			set_normalized_numeric(result,
								   datum_a.value + datum_b.value,
								   datum_a.weight);
		}
	}
	return true;
}

PUBLIC_FUNCTION(bool)
pgfn_numeric_sub(XPU_PGFUNCTION_ARGS)
{
	xpu_numeric_t  *result = (xpu_numeric_t *)__result;
	xpu_numeric_t	datum_a;
	xpu_numeric_t	datum_b;
	const kern_expression *karg = KEXP_FIRST_ARG(kexp);

	assert(kexp->nr_args == 2 &&
		   KEXP_IS_VALID(karg, numeric));
	if (!EXEC_KERN_EXPRESSION(kcxt, karg, &datum_a))
		return false;
	karg = KEXP_NEXT_ARG(karg);
	assert(KEXP_IS_VALID(karg, numeric));
	if (!EXEC_KERN_EXPRESSION(kcxt, karg, &datum_b))
		return false;
	if (XPU_DATUM_ISNULL(&datum_a) || XPU_DATUM_ISNULL(&datum_b))
		result->expr_ops = NULL;
	else
	{
		result->expr_ops = &xpu_numeric_ops;

		if (datum_a.kind != XPU_NUMERIC_KIND__VALID ||
			datum_b.kind != XPU_NUMERIC_KIND__VALID)
		{
			if (datum_a.kind == XPU_NUMERIC_KIND__NAN ||
				datum_b.kind == XPU_NUMERIC_KIND__NAN)
				result->kind = XPU_NUMERIC_KIND__NAN;
			else if (datum_a.kind == XPU_NUMERIC_KIND__POS_INF)
			{
				if (datum_b.kind == XPU_NUMERIC_KIND__POS_INF)
					result->kind = XPU_NUMERIC_KIND__NAN;	/* Inf - Inf */
				else
					result->kind = XPU_NUMERIC_KIND__POS_INF;
			}
			else if (datum_a.kind == XPU_NUMERIC_KIND__NEG_INF)
			{
				if (datum_b.kind == XPU_NUMERIC_KIND__NEG_INF)
					result->kind = XPU_NUMERIC_KIND__NAN;	/* -Inf - -Inf*/
				else
					result->kind = XPU_NUMERIC_KIND__NEG_INF;
			}
			else if (datum_b.kind == XPU_NUMERIC_KIND__POS_INF)
				result->kind = XPU_NUMERIC_KIND__NEG_INF;
			else
				result->kind = XPU_NUMERIC_KIND__POS_INF;
		}
		else
		{
			while (datum_a.weight > datum_b.weight)
			{
				datum_b.value *= 10;
				datum_b.weight++;
			}
			while (datum_a.weight < datum_b.weight)
			{
				datum_a.value *= 10;
				datum_a.weight++;
			}
			set_normalized_numeric(result,
								   datum_a.value - datum_b.value,
								   datum_a.weight);
		}
	}
	return true;
}

PUBLIC_FUNCTION(bool)
pgfn_numeric_mul(XPU_PGFUNCTION_ARGS)
{
	xpu_numeric_t  *result = (xpu_numeric_t *)__result;
	xpu_numeric_t	datum_a;
	xpu_numeric_t	datum_b;
	const kern_expression *karg = KEXP_FIRST_ARG(kexp);

	assert(kexp->nr_args == 2 &&
		   KEXP_IS_VALID(karg, numeric));
	if (!EXEC_KERN_EXPRESSION(kcxt, karg, &datum_a))
		return false;
	karg = KEXP_NEXT_ARG(karg);
	assert(KEXP_IS_VALID(karg, numeric));
	if (!EXEC_KERN_EXPRESSION(kcxt, karg, &datum_b))
		return false;
	if (XPU_DATUM_ISNULL(&datum_a) || XPU_DATUM_ISNULL(&datum_b))
		result->expr_ops = NULL;
	else
	{
		result->expr_ops = &xpu_numeric_ops;

		if (datum_a.kind != XPU_NUMERIC_KIND__VALID ||
			datum_b.kind != XPU_NUMERIC_KIND__VALID)
		{
			if (datum_a.kind == XPU_NUMERIC_KIND__NAN ||
				datum_a.kind == XPU_NUMERIC_KIND__NAN)
			{
				result->kind = XPU_NUMERIC_KIND__NAN;
			}
			else if (datum_a.kind == XPU_NUMERIC_KIND__POS_INF)
			{
				int		__sign = xpu_numeric_sign(&datum_b);

				if (__sign < 0)
					result->kind = XPU_NUMERIC_KIND__NEG_INF;
				else if (__sign > 0)
					result->kind = XPU_NUMERIC_KIND__POS_INF;
				else
					result->kind = XPU_NUMERIC_KIND__NAN;
			}
			else if (datum_a.kind == XPU_NUMERIC_KIND__NEG_INF)
			{
				int		__sign = xpu_numeric_sign(&datum_b);

				if (__sign < 0)
					result->kind = XPU_NUMERIC_KIND__POS_INF;
				else if (__sign > 0)
					result->kind = XPU_NUMERIC_KIND__NEG_INF;
				else
					result->kind = XPU_NUMERIC_KIND__NAN;
			}
			else if (datum_b.kind == XPU_NUMERIC_KIND__POS_INF)
			{
				int		__sign = xpu_numeric_sign(&datum_a);

				if (__sign < 0)
					result->kind = XPU_NUMERIC_KIND__NEG_INF;
				else if (__sign > 0)
					result->kind = XPU_NUMERIC_KIND__POS_INF;
				else
					result->kind = XPU_NUMERIC_KIND__NAN;
			}
			else
			{
				int		__sign = xpu_numeric_sign(&datum_a);

				if (__sign < 0)
					result->kind = XPU_NUMERIC_KIND__POS_INF;
				else if (__sign > 0)
					result->kind = XPU_NUMERIC_KIND__NEG_INF;
				else
					result->kind = XPU_NUMERIC_KIND__NAN;
			}
		}
		else
		{
			set_normalized_numeric(result,
								   datum_a.value * datum_b.value,
								   datum_a.weight + datum_b.weight);
		}
	}
	return true;
}

PUBLIC_FUNCTION(bool)
pgfn_numeric_div(XPU_PGFUNCTION_ARGS)
{
	xpu_numeric_t  *result = (xpu_numeric_t *)__result;
	xpu_numeric_t	datum_a;
	xpu_numeric_t	datum_b;
	const kern_expression *karg = KEXP_FIRST_ARG(kexp);

	assert(kexp->nr_args == 2 &&
		   KEXP_IS_VALID(karg, numeric));
	if (!EXEC_KERN_EXPRESSION(kcxt, karg, &datum_a))
		return false;
	karg = KEXP_NEXT_ARG(karg);
	assert(KEXP_IS_VALID(karg, numeric));
	if (!EXEC_KERN_EXPRESSION(kcxt, karg, &datum_b))
		return false;
	if (XPU_DATUM_ISNULL(&datum_a) || XPU_DATUM_ISNULL(&datum_b))
		result->expr_ops = NULL;
	else
	{
		result->expr_ops = &xpu_numeric_ops;

		if (datum_a.kind != XPU_NUMERIC_KIND__VALID ||
			datum_b.kind != XPU_NUMERIC_KIND__VALID)
		{
			if (datum_a.kind == XPU_NUMERIC_KIND__NAN ||
				datum_b.kind == XPU_NUMERIC_KIND__NAN)
			{
				result->kind = XPU_NUMERIC_KIND__NAN;
			}
			else if (datum_a.kind == XPU_NUMERIC_KIND__POS_INF)
			{
				int		__sign = xpu_numeric_sign(&datum_b);
				if (__sign == 1)
					result->kind = XPU_NUMERIC_KIND__NEG_INF;
				else if (__sign == -1)
					result->kind = XPU_NUMERIC_KIND__POS_INF;
				else if (__sign != 0)
					result->kind = XPU_NUMERIC_KIND__NAN;
				else
				{
					STROM_ELOG(kcxt, "division by zero");
					return false;
				}
			}
			else if (datum_a.kind == XPU_NUMERIC_KIND__NEG_INF)
			{
				int		__sign = xpu_numeric_sign(&datum_b);

				if (__sign == 1)
					result->kind = XPU_NUMERIC_KIND__NEG_INF;
				else if (__sign == -1)
					result->kind = XPU_NUMERIC_KIND__POS_INF;
				else if (__sign != 0)
					result->kind = XPU_NUMERIC_KIND__NAN;
				else
				{
					STROM_ELOG(kcxt, "division by zero");
					return false;
				}
			}
			else
			{
				/* by here, datum_a must be finite, so datum_b is not */
				set_normalized_numeric(result, 0, 0);
			}
		}
		else if (datum_b.value == 0)
		{
			STROM_ELOG(kcxt, "division by zero");
			return false;
		}
		else
		{
			int128_t	rem = datum_a.value;
			int128_t	div = datum_b.value;
			int128_t	x, ival = 0;
			int16_t		weight = datum_a.weight - datum_b.weight;
			bool		negative = false;

			if (rem < 0)
			{
				rem = -rem;
				if (div < 0)
					div = -div;
				else
					negative = true;
			}
			else if (div < 0)
			{
				negative = true;
				div = -div;
			}
			assert(rem >= 0 && div >= 0);

			for (;;)
			{
				x = rem / div;
				ival = 10 * ival + x;
				rem -= x * div;
				if (rem == 0)
					break;
				rem *= 10;
				weight++;
			}
			if (negative)
				ival = -ival;
			set_normalized_numeric(result, ival, weight);
		}
	}
	return true;
}

PUBLIC_FUNCTION(bool)
pgfn_numeric_mod(XPU_PGFUNCTION_ARGS)
{
	xpu_numeric_t  *result = (xpu_numeric_t *)__result;
	xpu_numeric_t	datum_a;
	xpu_numeric_t	datum_b;
	const kern_expression *karg = KEXP_FIRST_ARG(kexp);

	assert(kexp->nr_args == 2 &&
		   KEXP_IS_VALID(karg, numeric));
	if (!EXEC_KERN_EXPRESSION(kcxt, karg, &datum_a))
		return false;
	karg = KEXP_NEXT_ARG(karg);
	assert(KEXP_IS_VALID(karg, numeric));
	if (!EXEC_KERN_EXPRESSION(kcxt, karg, &datum_b))
		return false;
	if (XPU_DATUM_ISNULL(&datum_a) || XPU_DATUM_ISNULL(&datum_b))
		result->expr_ops = NULL;
	else
	{
		result->expr_ops = &xpu_numeric_ops;

		if (datum_a.kind != XPU_NUMERIC_KIND__VALID ||
			datum_b.kind != XPU_NUMERIC_KIND__VALID)
		{
			if (datum_a.kind == XPU_NUMERIC_KIND__NAN ||
				datum_b.kind == XPU_NUMERIC_KIND__NAN)
			{
				result->kind = XPU_NUMERIC_KIND__NAN;
			}
			else if (datum_a.kind == XPU_NUMERIC_KIND__POS_INF ||
					 datum_a.kind == XPU_NUMERIC_KIND__NEG_INF)
			{
				if (datum_b.kind == XPU_NUMERIC_KIND__VALID &&
					datum_b.value == 0)
				{
					STROM_ELOG(kcxt, "division by zero");
					return false;
				}
				else
				{
					result->kind = XPU_NUMERIC_KIND__NAN;
				}
			}
			else
			{
				/* num2 must be [-]Inf; result is num1 regardless of sign of num2 */
				result->kind = datum_b.kind;
				result->value = datum_b.value;
			}
		}
		else if (datum_b.value == 0)
		{
			STROM_ELOG(kcxt, "division by zero");
			return false;
		}
		else
		{
			while (datum_a.weight > datum_b.value)
			{
				datum_b.value *= 10;
				datum_b.weight++;
			}
			while (datum_a.weight < datum_b.weight)
			{
				datum_a.value *= 10;
				datum_a.weight++;
			}
			set_normalized_numeric(result,
								   datum_a.value % datum_b.value,
								   datum_a.weight);
		}
	}
	return true;
}

PUBLIC_FUNCTION(bool)
pgfn_numeric_uplus(XPU_PGFUNCTION_ARGS)
{
	const kern_expression *karg = KEXP_FIRST_ARG(kexp);

	assert(kexp->nr_args == 1 &&
		   KEXP_IS_VALID(karg, numeric));
	return EXEC_KERN_EXPRESSION(kcxt, karg, __result);
}

PUBLIC_FUNCTION(bool)
pgfn_numeric_uminus(XPU_PGFUNCTION_ARGS)
{
	xpu_numeric_t  *result = (xpu_numeric_t *)__result;
	const kern_expression *karg = KEXP_FIRST_ARG(kexp);

	assert(kexp->nr_args == 1 &&
		   KEXP_IS_VALID(karg, numeric));

	if (!EXEC_KERN_EXPRESSION(kcxt, karg, result))
		return false;
	if (!XPU_DATUM_ISNULL(result))
	{
		if (result->kind == XPU_NUMERIC_KIND__VALID)
			result->value = -result->value;
	}
	return true;
}

PUBLIC_FUNCTION(bool)
pgfn_numeric_abs(XPU_PGFUNCTION_ARGS)
{
	xpu_numeric_t  *result = (xpu_numeric_t *)__result;
	const kern_expression *karg = KEXP_FIRST_ARG(kexp);

	assert(kexp->nr_args == 1 &&
		   KEXP_IS_VALID(karg, numeric));
	if (!EXEC_KERN_EXPRESSION(kcxt, karg, result))
		return false;
	if (!XPU_DATUM_ISNULL(result))
	{
		if (result->kind == XPU_NUMERIC_KIND__VALID &&
			result->value < 0)
			result->value = -result->value;
	}
	return true;
}

PUBLIC_FUNCTION(int)
pg_numeric_to_cstring(kern_context *kcxt,
					  varlena *numeric,
					  char *buf, char *endp)
{
	NumericChoice *nc = (NumericChoice *)VARDATA_ANY(numeric);
	uint32_t	nc_len = VARSIZE_ANY_EXHDR(numeric);
	uint16_t	n_head = __Fetch(&nc->n_header);
	int			ndigits = NUMERIC_NDIGITS(n_head, nc_len);
	int			weight = NUMERIC_WEIGHT(nc, n_head);
	int			sign = NUMERIC_SIGN(n_head);
	int			dscale = NUMERIC_DSCALE(nc, n_head);
	int			d;
	char	   *cp = buf;
	NumericDigit *n_data = NUMERIC_DIGITS(nc, n_head);
	NumericDigit  dig, d1 __attribute__ ((unused));

	if (sign == NUMERIC_NEG)
	{
		if (cp >= endp)
			return -1;
		*cp++ = '-';
	}
	/* Output all digits before the decimal point */
	if (weight < 0)
	{
		d = weight + 1;
		if (cp >= endp)
			return -1;
		*cp++ = '0';
	}
	else
	{
		for (d = 0; d <= weight; d++)
		{
			bool		putit __attribute__ ((unused)) = (d > 0);

			if (d < ndigits)
				dig = __Fetch(n_data + d);
			else
				dig = 0;
#if PG_DEC_DIGITS == 4
			d1 = dig / 1000;
			dig -= d1 * 1000;
			putit |= (d1 > 0);
			if (putit)
			{
				if (cp >= endp)
					return -1;
				*cp++ = d1 + '0';
			}
			d1 = dig / 100;
			dig -= d1 * 100;
			putit |= (d1 > 0);
			if (putit)
			{
				if (cp >= endp)
					return -1;
				*cp++ = d1 + '0';
			}
			d1 = dig / 10;
			dig -= d1 * 10;
			putit |= (d1 > 0);
			if (putit)
			{
				if (cp >= endp)
					return -1;
				*cp++ = d1 + '0';
			}
			*cp++ = dig + '0';
#elif PG_DEC_DIGITS == 2
			d1 = dig / 10;
			dig -= d1 * 10;
			if (d1 > 0 || d > 0)
			{
				if (cp >= endp)
					return -1;
				*cp++ = d1 + '0';
			}
			if (cp >= endp)
				return -1;
			*cp++ = dig + '0';
#elif PG_DEC_DIGITS == 1
			if (cp >= endp)
				return -1;
			*cp++ = dig + '0';
#else
#error unsupported NBASE
#endif
		}
	}

	if (dscale > 0)
	{
		char   *lastp = cp;

		if (cp >= endp)
			return -1;
		*cp++ = '.';
		lastp = cp + dscale;
		for (int i = 0; i < dscale; d++, i += PG_DEC_DIGITS)
		{
			if (d >= 0 && d < ndigits)
				dig = __Fetch(n_data + d);
			else
				dig = 0;
#if PG_DEC_DIGITS == 4
			if (cp + 4 > endp)
				return -1;
			d1 = dig / 1000;
			dig -= d1 * 1000;
			*cp++ = d1 + '0';
			d1 = dig / 100;
			dig -= d1 * 100;
			*cp++ = d1 + '0';
			d1 = dig / 10;
			dig -= d1 * 10;
			*cp++ = d1 + '0';
			*cp++ = dig + '0';
			if (dig != 0)
				lastp = cp;
#elif PG_DEC_DIGITS == 2
			if (cp + 2 > endp)
				return -1;
			d1 = dig / 10;
			dig -= d1 * 10;
			*cp++ = d1 + '0';
			*cp++ = dig + '0';
#elif PG_DEC_DIGITS == 1
			if (cp >= endp)
				return -1;
			*cp++ = dig + '0';
#else
#error unsupported NBASE
#endif
			cp = lastp;
		}
	}
	return (int)(cp - buf);
}
