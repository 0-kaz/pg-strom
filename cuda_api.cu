/*
 * cuda_api.cu
 *
 * A set of simple wrappers to CUDA runtime APIs, because some of basic
 * declarations are conflicts between PostgreSQL and CUDA, thus we need
 * to invoke CUDA runtime API from files that does not include any
 * header files of PostgreSQL. That's too bad. :(
 *
 * --
 * Copyright 2011-2012 (c) KaiGai Kohei <kaigai@kaigai.gr.jp>
 *
 * This program is free software; you can redistribute it and/or
 * modify it under the terms of the 'LICENSE' included within
 * this package.
 */
#include "pg_strom_cuda.h"
#include <hip/hip_runtime.h>

const char *
pgcuda_get_error_string(hipError_t error)
{
	return hipGetErrorString(error);
}

hipError_t
pgcuda_get_device_count(int *count)
{
	return hipGetDeviceCount(count);
}	

hipError_t
pgcuda_set_device(int device)
{
	return hipSetDevice(device);
}

hipError_t
pgcuda_get_device(int *device)
{
	return hipGetDevice(device);
}

hipError_t
pgcuda_get_device_properties(struct hipDeviceProp_t *prop, int device)
{
	return hipGetDeviceProperties(prop, device);
}

hipError_t
pgcuda_malloc(void **devptr, size_t size)
{
	return hipMalloc(devptr, size);
}

hipError_t
pgcuda_free(void *devptr)
{
	return pgcuda_free(devptr);
}

hipError_t
pgcuda_malloc_host(void **ptr, size_t size)
{
	return hipHostMalloc(ptr, size);
}

hipError_t
pgcuda_free_host(void *ptr)
{
	return hipHostFree(ptr);
}

hipError_t
pgcuda_memcpy(void *dst, const void *src, size_t count,
			  enum hipMemcpyKind kind)
{
	return hipMemcpy(dst, src, count, kind);
}

hipError_t
pgcuda_memcpy_async(void *dst, const void *src, size_t count,
					enum hipMemcpyKind kind, hipStream_t stream)
{
	return hipMemcpyAsync(dst, src, count, kind, stream);
}

hipError_t
pgcuda_stream_create(hipStream_t *p_stream)
{
	return hipStreamCreate(p_stream);
}

hipError_t
pgcuda_stream_destroy(hipStream_t stream)
{
	return hipStreamDestroy(stream);
}

hipError_t
pgcuda_stream_synchronize(hipStream_t stream)
{
	return hipStreamSynchronize(stream);
}
